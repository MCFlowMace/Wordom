
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h> 
#include <stdlib.h>

__global__ void normalize(float a[3])
{
  float  b;

  b = sqrtf((float)(a[0]*a[0] + a[1]*a[1] + a[2]*a[2]));
  a[0] /= b;
  a[1] /= b;
  a[2] /= b;
}



__global__ void cross(float a[3], float b[3], float c[3])
{
  a[0] = b[1]*c[2] - b[2]*c[1];
  a[1] = b[2]*c[0] - b[0]*c[2];
  a[2] = b[0]*c[1] - b[1]*c[0];
}

__global__ void dot(float a[3], float b[3],float* c)
{
  //return (a[0] * b[0] + a[1] * b[1] + a[2] * b[2]);
  *c = a[0] * b[0] + a[1] * b[1] + a[2] * b[2];
}



/*
 * setup_rotation() 
 *
 *      given two lists of x,y,z coordinates, constructs
 * the correlation R matrix and the E value needed to calculate the
 * least-squares rotation matrix.
 */
__global__ void setup_rotation(const float *ref_xlist,
                    const float *mov_xlist, 
                    int& n_list,
                    float R[3][3],
                    float* E0)
{
  int i, j, n;

  /* initialize */
  for (i=0; i<3; i++)
    for (j=0; j<3; j++) 
      R[i][j] = 0.0;
  *E0 = 0.0;

  for (n=0; n<n_list; n++) 
  {
    /* 
     * E0 = 1/2 * sum(over n): y(n)*y(n) + x(n)*x(n) 
     */
    for (i=0; i<3; i++)
      *E0 +=  mov_xlist[3*n+i] * mov_xlist[3*n+i]  
            + ref_xlist[3*n+i] * ref_xlist[3*n+i];
    
    /*
     * correlation matrix R:   
     *   R[i,j) = sum(over n): y(n,i) * x(n,j)  
     *   where x(n) and y(n) are two vector sets   
     */
    for (i=0; i<3; i++)
    {
      for (j=0; j<3; j++)
        R[i][j] += mov_xlist[3*n+i] * ref_xlist[3*n+j];
    }
  }
  *E0 *= 0.5;
  }



#define ROTATE(a,i,j,k,l) { g = a[i][j]; \
                            h = a[k][l]; \
                            a[i][j] = g-s*(h+g*tau); \
                            a[k][l] = h+s*(g-h*tau); }
/*   
 * jacobi3
 *
 *    computes eigenval and eigen_vec of a real 3x3
 * symmetric matrix. On output, elements of a that are above 
 * the diagonal are destroyed. d[1..3] returns the 
 * eigenval of a. v[1..3][1..3] is a matrix whose 
 * columns contain, on output, the normalized eigen_vec of
 * a. n_rot returns the number of Jacobi rotations that were required.
 */
//__device__ int jacobi3(float a[3][3], float d[3], float v[3][3], int* n_rot)
 

__global__ void jacobiTest(float a[3][3], float d[3], float v[3][3])
{
  int count, k, i, j;
  float tresh, theta, tau, t, sum, s, h, g, c, b[3], z[3];

  /*Initialize v to the identity matrix.*/
  for (i=0; i<3; i++) 
  { 
    for (j=0; j<3; j++) 
      v[i][j] = 0.0;
    v[i][i] = 1.0;
  }

  /* Initialize b and d to the diagonal of a */
  for (i=0; i<3; i++) 
    b[i] = d[i] = a[i][i];

  /* z will accumulate terms */
  for (i=0; i<3; i++) 
    z[i] = 0.0; 
  
  //*n_rot = 0;

  /* 50 tries */
  for (count=0; count<50; count++)     
  {

    /* sum off-diagonal elements */
    sum = 0.0;
    for (i=0; i<2; i++) 
    {
      for (j=i+1; j<3; j++)
         sum += fabsf(a[i][j]);
    }

    /* if converged to machine underflow */
    if (sum == 0.0) 
      return;

    /* on 1st three sweeps... */
    if (count < 3) 
      tresh = sum * 0.2 / 9.0;    
    else       
      tresh = 0.0;      

    for (i=0; i<2; i++) 
    {
      for (j=i+1; j<3; j++) 
      {
        g = 100.0 * fabsf(a[i][j]);

        /*  after four sweeps, skip the rotation if
         *   the off-diagonal element is small 
         */
        if ( count > 3  &&  fabsf(d[i])+g == fabsf(d[i])
              &&  fabsf(d[j])+g == fabsf(d[j]) ) 
        {
          a[i][j] = 0.0;
        } 
        else if (fabsf(a[i][j]) > tresh) 
        {
          h = d[j] - d[i];
          
          if (fabsf(h)+g == fabsf(h))
          {
            t = a[i][j] / h;
          }
          else 
          {
            theta = 0.5 * h / (a[i][j]);
            t = 1.0 / ( fabsf(theta) +
                        (float)sqrtf(1.0 + theta*theta) );
            if (theta < 0.0) 
              t = -t;
          }
          
          c = 1.0 / (float) sqrtf(1 + t*t);
          s = t * c;
          tau = s / (1.0 + c);
          h = t * a[i][j];

          z[i] -= h;
          z[j] += h;
          d[i] -= h;
          d[j] += h;

          a[i][j] = 0.0;

         /* for (k=0; k<=i-1; k++) 
            ROTATE(a, k, i, k, j)

          for (k=i+1; k<=j-1; k++) 
            ROTATE(a, i, k, k, j)

          for (k=j+1; k<3; k++) 
            ROTATE(a, i, k, j, k)

          for (k=0; k<3; k++) 
            ROTATE(v, k, i, k, j)*/

          //++(*n_rot);
        }
      }
    }

    for (i=0; i<3; i++) 
    {
      b[i] += z[i];
      d[i] = b[i];
      z[i] = 0.0;
    }
  }

  printf("Too many iterations in jacobi3\n");
  //return (0);
}  



/* 
 * diagonalize_symmetric 
 *
 *    Diagonalize a 3x3 matrix & sort eigenval by size
 */

__global__ void diagonalize_Test(float matrix[3][3], float vec[3][3], float eigenval[3])
//__device__ int diagonalize_symmetric(float matrix[3][3], float eigenval[3])
{
  //int n_rot, i, j, k;
  int i, j, k;
  //float vec[3][3];
  float val; 
  
  //if (!jacobi3(matrix, eigenval, vec, &n_rot))
 /* if (!jacobi3(matrix, eigenval, vec))
  {
    //printf("convergence failed\n");
    return ;
  }*/

  /* sort solutions by eigenval */
  for (i=0; i<3; i++) 
  {
    k = i;
    val = eigenval[i];
    
    for (j=i+1; j<3; j++)
      if (eigenval[j] >= val)
      { 
        k = j;
        val = eigenval[k];
      }
       
    if (k != i) 
    {
      eigenval[k] = eigenval[i];
      eigenval[i] = val;
      for (j=0; j<3; j++) 
      {
        val = vec[j][i];
        vec[j][i] = vec[j][k];
        vec[j][k] = val;
      }
    }
  }

  /* transpose such that first index refers to solution index */
  for (i=0; i<3; i++)
    for (j=0; j<3; j++)
      vec[i][j] = vec[j][i];

  //return (1);
}



/*
 * calculate_rotation_matrix() 
 *
 *   calculates the rotation matrix U and the
 * rmsd from the R matrix and E0:
 */
__global__ void calculate_rotation_matrix(float R[3][3],
                              float& E0,
                              float* residual)
{
  int i, j, k;
  float Rt[3][3], RtR[3][3];
  //float RtR[3][3];
  //float eigenval[3];
  float left_eigenvec[3][3], right_eigenvec[3][3], eigenval[3];
  float v[3];
  float sigma;

  /* build Rt, transpose of R  */
  for (i=0; i<3; i++)
    for (j=0; j<3; j++)
      Rt[i][j] = R[j][i];

  /* make symmetric RtR = Rt X R */
  for (i=0; i<3; i++) 
    for (j=0; j<3; j++)
    {
      RtR[i][j] = 0.0;
      for (k = 0; k<3; k++)
        RtR[i][j] += Rt[k][i] * R[j][k];
        //RtR[i][j] += R[i][k] * R[j][k];
    }

  //if (!diagonalize_symmetric(RtR, right_eigenvec, eigenval))
    //return(0);

  // right_eigenvec's should be an orthogonal system but could be left
  // or right-handed. Let's force into right-handed system.
   
 // cross(&right_eigenvec[2][0], &right_eigenvec[0][0], &right_eigenvec[1][0]);

  // From the Kabsch algorithm, the eigenvec's of RtR
  // are identical to the right_eigenvec's of R.
  // This means that left_eigenvec = R x right_eigenvec 
   
  /*for (i=0; i<3; i++) 
    for (j=0; j<3; j++) 
      left_eigenvec[i][j] = dot(&right_eigenvec[i][0], &Rt[j][0]);*/
      //left_eigenvec[i][j] = dot(&right_eigenvec[i][0], &R[0][j]);

  //for (i=0; i<3; i++) 
   // normalize(&left_eigenvec[i][0]);

   
   // Force left_eigenvec[2] to be orthogonal to the other vectors.
   // First check if the rotational matrices generated from the 
   // orthogonal eigenvectors are in a right-handed or left-handed
   // co-ordinate system - given by sigma. Sigma is needed to
   // resolve this ambiguity in calculating the RMSD.
   
  /*cross(v, &left_eigenvec[0][0], &left_eigenvec[1][0]);
  if (dot(v, &left_eigenvec[2][0]) < 0.0)
    sigma = -1.0;
  else 
    sigma = 1.0;*/
  for (i=0; i<3; i++)
    left_eigenvec[2][i] = v[i];

    
  *residual = E0 - (float) sqrt(fabs(eigenval[0])) 
                 - (float) sqrt(fabs(eigenval[1]))
                 - sigma * (float) sqrt(fabs(eigenval[2]));

  //return (1);
}

