#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h> 
#include <stdlib.h>
	//including wordom.h generates compiler warnings
	//we do not have to worry about this, since this only happens because nvcc generates c++ object files and the c++ compiler is not completely satisfied with some stuff in wordom.h
#include "wordom.h"
#include "fileio.h"
#include "tools.h"
#include "analysis.h"
#include "cluster.h"

__device__ void normalize(float a[3])
{
  float  b;

  b = sqrtf((float)(a[0]*a[0] + a[1]*a[1] + a[2]*a[2]));
  a[0] /= b;
  a[1] /= b;
  a[2] /= b;
}



__device__ float dot(float a[3], float b[3])
{
  return (a[0] * b[0] + a[1] * b[1] + a[2] * b[2]);
}



__device__ static void cross(float a[3], float b[3], float c[3])
{
  a[0] = b[1]*c[2] - b[2]*c[1];
  a[1] = b[2]*c[0] - b[0]*c[2];
  a[2] = b[0]*c[1] - b[1]*c[0];
}



/*
 * setup_rotation() 
 *
 *      given two lists of x,y,z coordinates, constructs
 * the correlation R matrix and the E value needed to calculate the
 * least-squares rotation matrix.
 */
__device__ void setup_rotation(const float *ref_xlist, const float *mov_xlist, int& n_list, float R[3][3], float* E0, volatile int& i, volatile int&j, volatile int& n)
{
  //int i, j, n;

  /* initialize */
  for (i=0; i<3; i++)
    for (j=0; j<3; j++) 
      R[i][j] = 0.0;
  *E0 = 0.0;

  for (n=0; n<n_list; n++) 
  {
    /* 
     * E0 = 1/2 * sum(over n): y(n)*y(n) + x(n)*x(n) 
     */
    for (i=0; i<3; i++)
      *E0 +=  mov_xlist[3*n+i] * mov_xlist[3*n+i]  
            + ref_xlist[3*n+i] * ref_xlist[3*n+i];
    
    /*
     * correlation matrix R:   
     *   R[i,j) = sum(over n): y(n,i) * x(n,j)  
     *   where x(n) and y(n) are two vector sets   
     */
    for (i=0; i<3; i++)
    {
      for (j=0; j<3; j++)
        R[i][j] += mov_xlist[3*n+i] * ref_xlist[3*n+j];
    }
  }
  *E0 *= 0.5;
  }



#define ROTATE(a,i,j,k,l) { g = a[i][j]; \
                            h = a[k][l]; \
                            a[i][j] = g-s*(h+g*tau); \
                            a[k][l] = h+s*(g-h*tau); }
/*   
 * jacobi3
 *
 *    computes eigenval and eigen_vec of a real 3x3
 * symmetric matrix. On output, elements of a that are above 
 * the diagonal are destroyed. d[1..3] returns the 
 * eigenval of a. v[1..3][1..3] is a matrix whose 
 * columns contain, on output, the normalized eigen_vec of
 * a. n_rot returns the number of Jacobi rotations that were required.
 */
__device__ int jacobi3(float a[3][3], float d[3], float v[3][3], volatile int& k, volatile int& i, volatile int& j)
{
  //int count, k, i, j;
  //float tresh, theta, tau, t, sum, s, h, g, c, b[3], z[3];
  //float t, sum, s, h, g, c, b[3], z[3];
  float b[3], z[3];

  /*Initialize v to the identity matrix.*/
  for (i=0; i<3; i++) 
  { 
    for (j=0; j<3; j++) 
      v[i][j] = 0.0;
    v[i][i] = 1.0;
  }

  /* Initialize b and d to the diagonal of a */
  for (i=0; i<3; i++) 
    b[i] = d[i] = a[i][i];

  /* z will accumulate terms */
  for (i=0; i<3; i++) 
    z[i] = 0.0; 
  
  //*n_rot = 0;

  /* 50 tries */
  int count;
  for (count=0; count<50; count++)     
  {
	
	float tresh;
    /* sum off-diagonal elements */
    {
    float sum = 0.0;
    for (i=0; i<2; i++) 
    {
      for (j=i+1; j<3; j++)
         sum += fabsf(a[i][j]);
    }

    /* if converged to machine underflow */
    if (sum == 0.0) 
      return(1);

    /* on 1st three sweeps... */
    if (count < 3) 
      tresh = sum * 0.2 / 9.0;    
    else       
      tresh = 0.0;
      
  } 

    for (i=0; i<2; i++) 
    {
      for (j=i+1; j<3; j++) 
      {
        float g = 100.0 * fabsf(a[i][j]);

        /*  after four sweeps, skip the rotation if
         *   the off-diagonal element is small 
         */
       if ( count > 3  &&  fabsf(d[i])+g == fabsf(d[i])
              &&  fabsf(d[j])+g == fabsf(d[j]) ) 
        {
          a[i][j] = 0.0;
        } 
        else if (fabsf(a[i][j]) > tresh) 
        {
          float h = d[j] - d[i];
          float t;
          
          if (fabsf(h)+g == fabsf(h))
          {
            t = a[i][j] / h;
          }
          else 
          {
            float theta = 0.5 * h / (a[i][j]);
            t = 1.0 / ( fabsf(theta) +
                        (float)sqrtf(1.0 + theta*theta) );
            if (theta < 0.0) 
              t = -t;
          }
          
          float c = 1.0 / (float) sqrtf(1 + t*t);
          float s = t * c;
          float tau = s / (1.0 + c);
          h = t * a[i][j];

          z[i] -= h;
          z[j] += h;
          d[i] -= h;
          d[j] += h;

          a[i][j] = 0.0;

          for (k=0; k<=i-1; k++) 
            ROTATE(a, k, i, k, j)

          for (k=i+1; k<=j-1; k++) 
            ROTATE(a, i, k, k, j)

          for (k=j+1; k<3; k++) 
            ROTATE(a, i, k, j, k)

          for (k=0; k<3; k++) 
            ROTATE(v, k, i, k, j)

         // ++(*n_rot);
        }
      }
    }

    for (i=0; i<3; i++) 
    {
      b[i] += z[i];
      d[i] = b[i];
      z[i] = 0.0;
    }
  }

  printf("Too many iterations in jacobi3\n");
  return (0);
}  



/* 
 * diagonalize_symmetric 
 *
 *    Diagonalize a 3x3 matrix & sort eigenval by size
 */
__device__ int diagonalize_symmetric(float matrix[3][3], float vec[3][3], float eigenval[3], volatile int& i, volatile int& j, volatile int& k)
{
  //int n_rot, i, j, k;
  //float vec[3][3];
  //float val; 
  
  if (!jacobi3(matrix, eigenval, vec, i,j,k)) 
  {
    //printf("convergence failed\n");
    return (0);
  }

  /* sort solutions by eigenval */
  for (i=0; i<3; i++) 
  {
	  float val;
    k = i;
    val = eigenval[i];
    
    for (j=i+1; j<3; j++)
      if (eigenval[j] >= val)
      { 
        k = j;
        val = eigenval[k];
      }
       
    if (k != i) 
    {
      eigenval[k] = eigenval[i];
      eigenval[i] = val;
      for (j=0; j<3; j++) 
      {
        val = vec[j][i];
        vec[j][i] = vec[j][k];
        vec[j][k] = val;
      }
    }
  }

  /* transpose such that first index refers to solution index */
  for (i=0; i<3; i++)
    for (j=0; j<3; j++)
      vec[i][j] = vec[j][i];

  return (1);
}



/*
 * calculate_rotation_matrix() 
 *
 *   calculates the rotation matrix U and the
 * rmsd from the R matrix and E0:
 */
//__device__ int calculate_rotation_matrix(float R[3][3], float E0, float* residual, volatile int& i, volatile int& j, volatile int& k)
__device__ int calculate_rotation_matrix(float R[3][3], float& residual, volatile int& i, volatile int& j, volatile int& k)
{
  //int i, j, k;
  float Rt[3][3];
  float right_eigenvec[3][3], eigenval[3];
  //float v[3];
  //float sigma;

  /* build Rt, transpose of R  */
  for (i=0; i<3; i++)
    for (j=0; j<3; j++)
      Rt[i][j] = R[j][i];

  /* make symmetric RtR = Rt X R */
  
  {
  float RtR[3][3];
  for (i=0; i<3; i++) 
    for (j=0; j<3; j++)
    {
      RtR[i][j] = 0.0;
      for (k = 0; k<3; k++)
        RtR[i][j] += Rt[k][i] * R[j][k];
    }

  if (!diagonalize_symmetric(RtR, right_eigenvec, eigenval,i,j,k))
    return(0);
  }

  /* right_eigenvec's should be an orthogonal system but could be left
   * or right-handed. Let's force into right-handed system.
   */
  cross(&right_eigenvec[2][0], &right_eigenvec[0][0], &right_eigenvec[1][0]);

  /* From the Kabsch algorithm, the eigenvec's of RtR
   * are identical to the right_eigenvec's of R.
   * This means that left_eigenvec = R x right_eigenvec 
   */
   
   float left_eigenvec[3][3];
  for (i=0; i<3; i++) 
    for (j=0; j<3; j++) 
      left_eigenvec[i][j] = dot(&right_eigenvec[i][0], &Rt[j][0]);

  for (i=0; i<3; i++) 
    normalize(&left_eigenvec[i][0]);

  /* 
   * Force left_eigenvec[2] to be orthogonal to the other vectors.
   * First check if the rotational matrices generated from the 
   * orthogonal eigenvectors are in a right-handed or left-handed
   * co-ordinate system - given by sigma. Sigma is needed to
   * resolve this ambiguity in calculating the RMSD.
   */
  //cross(v, &left_eigenvec[0][0], &left_eigenvec[1][0]);
  cross(&right_eigenvec[0][0], &left_eigenvec[0][0], &left_eigenvec[1][0]);
  
  //if (dot(v, &left_eigenvec[2][0]) < 0.0)
  float sigma;
  if (dot(&right_eigenvec[0][0], &left_eigenvec[2][0]) < 0.0)
    sigma = -1.0;
  else 
    sigma = 1.0;
    
  //for (i=0; i<3; i++)
    //left_eigenvec[2][i] = v[i]; 

    
  //*residual = E0 - (float) sqrtf(fabsf(eigenval[0])) - (float) sqrtf(fabsf(eigenval[1]))- sigma * (float) sqrtf(fabsf(eigenval[2]));
  residual = residual - (float) sqrtf(fabsf(eigenval[0])) - (float) sqrtf(fabsf(eigenval[1]))- sigma * (float) sqrtf(fabsf(eigenval[2]));
                 

  return (1);
}



/*__device__ void calculate_rotation_rmsd(const float* ref_xlist,
                             const float* mov_xlist, 
                             int n_list,
                             float* rmsd)
{
  float Eo, residual;
  float R[3][3];
  
  setup_rotation(ref_xlist, mov_xlist, n_list, R, &Eo);
                 
  if(calculate_rotation_matrix(R, Eo, &residual)) {
  
		residual = fabsf(residual); // avoids the awkward case of -0.0 
		*rmsd = sqrtf( fabsf((float) (residual)*2.0/((float)n_list)) ); 
  
	}               
                 
}*/
 
 

__device__ float RmsdCalc_nosup(const float *refcoor, const float *movcoor, int nato)
{
  // compute rmsd and return value
  int          ii;
  float        rmsd,di;
  
  rmsd=0;
  di=0;
  
  for ( ii=0; ii<3*nato; ii++ ) {
		di= refcoor[ii]-movcoor[ii];
		rmsd += di*di;
             
	}
  
  rmsd /= nato;
  rmsd = sqrt ( rmsd );
  
  return rmsd;
}


/*__device__ float RmsdCalc(const float *refcoor, const float *movcoor, int nato, int super )
{
  float rmsd=-1;
  
  if( super == 0 )
    return RmsdCalc_nosup( refcoor, movcoor, nato );
  
  calculate_rotation_rmsd(refcoor,movcoor, nato, &rmsd);
  
  return rmsd;
  
}*/

__global__ void gRmsdMaxSuper(int nato, const int nframes, const float* gclust_coords, const int cluster, const float cutoff, const int* frameapp_read, int* frameapp_write, float* distance) {
	
	
	// nato = number of atoms, nframes = number of frames used for clustering;
	// gclust_coords = array of all coordinates of the frames; cluster = center of a possible cluster; frameapp stores the cluster center 
	// of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays;
	// distance stores the calculated rmsd because it will be needed in the post processing
	

	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	//with the maxspeed flag frames are assigned to the first cluster -> do not check frame again if it was already assigned
	if(frameapp_read[index] != -1) {
		frameapp_write[index] = frameapp_read[index];
		return;
	}
	
	//if cluster has already been assigned to another cluster then it cannot be a new cluster
	if(frameapp_read[cluster] == -1) {

		//spare the distance calculation if index and cluster are the same, by wordoms conventions the distance is set to -1 instead of 0 in this case
		if(cluster == index) {
			frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = -1.0;
			return;
		}
		
		//float rmsd=-1;
	
		//calculate_rotation_rmsd(&gclust_coords[cluster*3*nato],&gclust_coords[index*3*nato], nato, &rmsd);
		
				//calculate_rotation_rmsd(&gclust_coords[cluster*3*nato],&gclust_coords[index*3*nato], nato, &rmsd);
		float rmsd;
		float R[3][3];
		
		//volatile int i,j,k,n;
		volatile int i,j,n;	
		setup_rotation(&gclust_coords[cluster*3*nato],&gclust_coords[index*3*nato], nato, R, &rmsd, i, j, n);
		
		//setup_rotation(&gclust_coords[cluster*3*nato],&gclust_coords[index*3*nato], nato, R, &E0);		
			
		//float rmsd=-1;
		//if(calculate_rotation_matrix(R, E0, &rmsd, i, j, n)) {
		if(calculate_rotation_matrix(R, rmsd, i, j, n)) {
		//if(calculate_rotation_matrix(R, E0, &rmsd)) {
  
			rmsd = fabsf(rmsd); // avoids the awkward case of -0.0 
			rmsd = sqrtf( fabsf((float) (rmsd)*2.0/((float)nato)) ); 
			
			if (rmsd<cutoff){
				frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
				distance[index] = rmsd;
			return;
			}
  
		}	
			
		/*if (rmsd<cutoff && rmsd != -1){
			frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = rmsd;
			return;
		}*/
		
	}
	frameapp_write[index] = frameapp_read[index];
			  
			
}

/*__global__ void gRmsdMax(const int nato, const int nframes, float* gclust_coordsX, float* gclust_coordsY, float* gclust_coordsZ, const int cluster, const float cutoff, const int* frameapp_read, int* frameapp_write, float* distance, const int super) {
	
	
	// nato = number of atoms, nframes = number of frames used for clustering;
	// gclust_coords = array of all coordinates of the frames; cluster = center of a possible cluster; frameapp stores the cluster center 
	// of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays;
	// distance stores the calculated rmsd because it will be needed in the post processing
	
	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	//if(index <100) printf("This works %d\n",index);
	
	//with the maxspeed flag frames are assigned to the first cluster -> do not check frame again if it was already assigned
	if(frameapp_read[index] != -1) {
		frameapp_write[index] = frameapp_read[index];
		return;
	}
	
	//if cluster has already been assigned to another cluster then it cannot be a new cluster
	if(frameapp_read[cluster] == -1) {
		
		//spare the distance calculation if index and cluster are the same, by wordoms conventions the distance is set to -1 instead of 0 in this case
		if(cluster == index) {
			frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = -1.0;
			return;
		}
		
		float *g3_index[3];
		float *g3_cluster[3];
		
		g3_index[0]=&gclust_coordsX[index*nato];
		g3_index[1]=&gclust_coordsY[index*nato];
		g3_index[2]=&gclust_coordsZ[index*nato];
		
		g3_cluster[0]=&gclust_coordsX[cluster*nato];
		g3_cluster[1]=&gclust_coordsY[cluster*nato];
		g3_cluster[2]=&gclust_coordsZ[cluster*nato];
		
		float rmsd;//=0.; , di;
		
		rmsd = RmsdCalc(g3_cluster,g3_index,nato,super);
		
		fast_rmsd(float ref_xlist[][3], float mov_xlist[][3], int n_list, float* rmsd)
		
		//int ii;
		
	    
	    // compute rmsd and return value
			  
		//for ( ii=0; ii<nato; ii++ ) {
			//rmsd += ( (gclust_coordsX[index*nato+ii]-gclust_coordsX[cluster*nato+ii])*(gclust_coordsX[index*nato+ii]-gclust_coordsX[cluster*nato+ii]) + 
             //(gclust_coordsY[index*nato+ii]-gclust_coordsY[cluster*nato+ii])*(gclust_coordsY[index*nato+ii]-gclust_coordsY[cluster*nato+ii]) + 
             //(gclust_coordsZ[index*nato+ii]-gclust_coordsZ[cluster*nato+ii])*(gclust_coordsZ[index*nato+ii]-gclust_coordsZ[cluster*nato+ii]) );
			//}
			  
		//rmsd /= nato;
		//rmsd = sqrt ( rmsd );
	
		if (rmsd<cutoff){		
			frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = rmsd;
			return;
		}
		
	}
	frameapp_write[index] = frameapp_read[index];
			  
			
}*/

__global__ void gRmsdClosest(const int nato, const int nframes, const float* gclust_coords, const int cluster, const int* frameapp_read, int* frameapp_write, float* distance) {
	
	// nato = number of atoms; nframes = number of frames used for clustering;
	// gclust_coords = array of all coordinates of the frames; cluster = center of a possible cluster; frameapp stores the cluster center 
	// of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays;
	// distance stores the calculated drms for comparison
	

	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	//frames can only be assigned to clusters with a lower frame number
	if(cluster > index) {
		frameapp_write[index] = frameapp_read[index];
		return;
	}
	
	//if cluster has already been assigned to another cluster then it cannot be a new cluster
	if(frameapp_read[cluster] == -1) {
	
		//spare the distance calculation if index and cluster are the same, by wordoms conventions the distance is set to -1 instead of 0 in this case
		if(cluster == index) {
			frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = -1.0;
			return;
		}		
		float rmsd=0. , di;
		int jj;
		
		//calculate the drms of cluster and index
		for (jj=0;jj<3*nato;jj++){
			di=(gclust_coords[index*3*nato+jj]-gclust_coords[cluster*3*nato+jj]);
			rmsd+=di*di; 
		}			
		rmsd /= nato;
		rmsd = sqrt ( rmsd );
	
		//at the beginning distance is set to the cutoff, by always comparing the drms to the current value of distance instead of only the cutoff we can reassign the frame if we find a closer cluster
		if (rmsd < distance[index]){		
			frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = rmsd;
			return;
		}
	}
	
	frameapp_write[index] = frameapp_read[index];
}


__global__ void gRmsdClustersMax(const int nato, const int nframes, const float* gclust_coords, const int cluster, const float cutoff, const int* frameapp_read, int* frameapp_write, float* distance, const int nclusters, const int clustercenter) {
	

	// nato = number of atoms; nframes = number of frames used for clustering;
	// gclust_coords = array of all coordinates of the frames; cluster = number of the cluster; frameapp stores the cluster center 
	// of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays; 
	// distance stores the calculated drms because it will be needed in the post processing;
	// nclusters passes the number of already found clusters; clustercenter passes the center of the current cluster

	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x; 	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	//with maxspeed frames are assigned to the first cluster -> do not check frame again if it was already assigned
	if(frameapp_read[index] != -1) {
		frameapp_write[index] = frameapp_read[index];
		return;
	}		

	float rmsd=0. , di;
	int ii;
	
	    // compute rmsd and return value			  

	for ( ii=0; ii<3*nato; ii++ ) {
			di=gclust_coords[(index+nclusters)*3*nato+ii]-gclust_coords[cluster*3*nato+ii];
			rmsd+=di*di;
			/*di=gclust_coords[nframes*nato+index*nato+ii]-gclust_coords[nframes*nato+cluster*nato+ii];
			rmsd+=di*di;
			di=gclust_coords[2*nframes*nato+index*nato+ii]-gclust_coords[2*nframes*nato+cluster*nato+ii];
			rmsd+=di*di;*/
		}
			  
		rmsd /= nato;
		rmsd = sqrt ( rmsd );

		if (rmsd<cutoff){		
			frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = rmsd;
			return;
		}
	
	frameapp_write[index] = frameapp_read[index];
}

__global__ void gRmsdFramesMax(const int nato, const int nframes, const float* gclust_coords, const int cluster, const float cutoff, const int* frameapp_read, int* frameapp_write, float* distance, const int framesFinished, const int nclusters, int* newClusters, int* clusterCenters) {
	
	
	// nato = number of atoms; nframes = number of frames used for clustering;
	// gclust_coords = array of all coordinates of the frames; cluster = center of a possible cluster; frameapp stores the cluster center 
	// of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays;
	// distance stores the calculated drms because it will be needed in the post processing;
	// framesFinished passes the number of processed frames in a previous chunk; nclusters passes the number of already found clusters;
	// newClusters stores the number of new found clusters in this chunk; clusterCenters stores the centers of these clusters
	

	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	//with maxspeed frames are assigned to the first cluster -> do not check frame again if it was already assigned
	if(frameapp_read[index] != -1) {
		frameapp_write[index] = frameapp_read[index];
		return;
	}
	
	//if cluster has already been assigned to another cluster then it cannot be a new cluster
	if(frameapp_read[cluster] == -1) {
		
		//spare the distance calculation if index and cluster are the same, by wordoms conventions the distance is set to -1 instead of 0 in this case
		if(cluster == index) {
			frameapp_write[index] = framesFinished + cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = -1.0;
			clusterCenters[*newClusters] = framesFinished + cluster + 1;
			(*newClusters)++;
			return;
		}
		
		float rmsd=0. , di;
		int jj;
		
		//calculate the drms of cluster and index
		for (jj=0;jj<3*nato;jj++){
			di=gclust_coords[(index+nclusters)*3*nato+jj]-gclust_coords[(cluster+nclusters)*3*nato+jj];
			rmsd+=di*di; 
		}
					
		rmsd /= nato;
		rmsd = sqrt ( rmsd );
	
		if (rmsd<cutoff){		
			frameapp_write[index] = framesFinished + cluster + 1;
			distance[index] = rmsd;
			return;
		}

	}
	frameapp_write[index] = frameapp_read[index];
}


__global__ void gRmsdClustersClosest(const int nato, const int nframes, const float* gclust_coords, const int cluster, const int* frameapp_read, int* frameapp_write, float* distance, const int nclusters, const int clustercenter) {
	
	// nato = number of atoms; nframes = number of frames used for clustering;
	// gclust_coords = array of all coordinates of the frames; cluster = number of the cluster; frameapp stores the cluster center 
	// of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays; 
	// distance stores the calculated for comparison;
	// nclusters passes the number of already found clusters; clustercenter passes the center of the current cluster
	

	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	float rmsd=0. , di;
	int jj;
	
	//calculate the rmsd of cluster and index
	for (jj=0;jj<3*nato;jj++){
		di=(gclust_coords[(index+nclusters)*3*nato+jj]-gclust_coords[cluster*3*nato+jj]);
		rmsd+=di*di; 
	}
		
	rmsd /= nato;
	rmsd = sqrt ( rmsd );

	//at the beginning distance is set to cutoff, by always comparing the drms to the current value of distance instead of the cutoff we can reassign the frame if we find a closer cluster
	if (rmsd < distance[index]){		
		frameapp_write[index] = clustercenter;
		distance[index] = rmsd;
		return;
	}
	frameapp_write[index] = frameapp_read[index];
	
}

__global__ void gRmsdFramesClosest(const int nato, const int nframes, const float* gclust_coords, const int cluster, const int* frameapp_read, int* frameapp_write, float* distance, const int framesFinished, const int nclusters, int* newClusters, int* clusterCenters) {
	
	// nato = number of atoms; nframes = number of frames used for clustering;
	// gclust_coords = array of all coordinates of the frames; cluster = center of a possible cluster; frameapp stores the cluster center 
	// of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays;
	// distance stores the calculated drms for comparison;
	// framesFinished passes the number of processed frames in a previous chunk; nclusters passes the number of already found clusters;
	// newClusters stores the number of new found clusters in this chunk; clusterCenters stores the centers of these clusters	
	
	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	//frames can only be assigned to clusters with a lower frame number
	if(cluster > index) {
		frameapp_write[index] = frameapp_read[index];
		return;
	}
	
	//if cluster has already been assigned to another cluster then it cannot be a new cluster
	if(frameapp_read[cluster] == -1) {
	
		//spare the distance calculation if index and cluster are the same, by wordoms conventions the distance is set to -1 instead of 0 in this case
		if(cluster == index) {
			frameapp_write[index] = framesFinished + cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = -1.0;
			clusterCenters[*newClusters] = framesFinished + cluster + 1;
			(*newClusters)++;
			return;
		}		
		float rmsd=0. , di;
		int jj;
		
		//calculate the drms of cluster and index
		for (jj=0;jj<3*nato;jj++){
			di=(gclust_coords[(index+nclusters)*3*nato+jj]-gclust_coords[(cluster+nclusters)*3*nato+jj]);
			rmsd+=di*di; 
		}			
		rmsd /= nato;
		rmsd = sqrt ( rmsd );
	
		//at the beginning distance is set to cutoff, by always comparing the drms to the current value of distance instead of the cutoff we can reassign the frame if we find a closer cluster
		if (rmsd < distance[index]){	
			frameapp_write[index] = framesFinished + cluster + 1;
			distance[index] = rmsd;
			return;
		}	
	}
	frameapp_write[index] = frameapp_read[index];
	
}

// a kernel for the maxspeed flag
__global__ void gDrmsMax(const int msize, const int nframes, const float* gclust_dmtx, const int cluster, const float cutoff, const int* frameapp_read, int* frameapp_write, float* distance, const float nointrasegm_corr_fact) {
	
	// msize = size of each distance matrix, nframes = number of frames used for clustering;
	// gclust_dmtx = array of ALL distance matrices; cluster = center of a possible cluster; frameapp stores the cluster center 
	// of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays;
	// distance stores the calculated drms because it will be needed in the post processing
	
	

	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	//with the maxspeed flag frames are assigned to the first cluster -> do not check frame again if it was already assigned
	if(frameapp_read[index] != -1) {
		frameapp_write[index] = frameapp_read[index];
		return;
	}
	
	//if cluster has already been assigned to another cluster then it cannot be a new cluster
	if(frameapp_read[cluster] == -1) {
		
		//spare the distance calculation if index and cluster are the same, by wordoms conventions the distance is set to -1 instead of 0 in this case
		if(cluster == index) {
			frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = -1.0;
			return;
		}
		
		
		float drms=0. , di;
		int jj;
		
		//calculate the drms of cluster and index
		for (jj=0;jj<msize;jj++){
			di=(gclust_dmtx[index*msize+jj]-gclust_dmtx[cluster*msize+jj]);
			drms+=di*di; 
		}		
		drms = sqrtf(drms);
	    drms *= nointrasegm_corr_fact/sqrtf((float)msize); //Renormalize the distance properly
	
		if (drms<cutoff){		
			frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = drms;
			return;
		}
		
	}
	frameapp_write[index] = frameapp_read[index];
}

// a kernel for the lfull flag
__global__ void gDrmsClosest(const int msize, const int nframes, const float* gclust_dmtx, const int cluster, const int* frameapp_read, int* frameapp_write, float* distance, const float nointrasegm_corr_fact) {
	
	// msize = size of each distance matrix; nframes = number of frames used for clustering;
	// gclust_dmtx = array of ALL distance matrices; cluster = center of a possible cluster; frameapp stores the cluster center 
	// of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays;
	// distance stores the calculated drms for comparison

	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	//frames can only be assigned to clusters with a lower frame number
	if(cluster > index) {
		frameapp_write[index] = frameapp_read[index];
		return;
	}
	
	//if cluster has already been assigned to another cluster then it cannot be a new cluster
	if(frameapp_read[cluster] == -1) {
	
		//spare the distance calculation if index and cluster are the same, by wordoms conventions the distance is set to -1 instead of 0 in this case
		if(cluster == index) {
			frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = -1.0;
			return;
		}		
		float drms=0. , di;
		int jj;
		
		//calculate the drms of cluster and index
		for (jj=0;jj<msize;jj++){
			di=(gclust_dmtx[index*msize+jj]-gclust_dmtx[cluster*msize+jj]);
			drms+=di*di; 
		}			
		drms = sqrtf(drms);
	    drms *= nointrasegm_corr_fact/sqrtf((float)msize); //Renormalize the distance properly
	
		//at the beginning distance is set to the cutoff, by always comparing the drms to the current value of distance instead of only the cutoff we can reassign the frame if we find a closer cluster
		if (drms < distance[index]){		
			frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = drms;
			return;
		}
	}
	
	frameapp_write[index] = frameapp_read[index];
}

// a kernel for the maxspeed flag for calculation with limited memory; this is for comparing the frames of the chunk to the previously found clusters
__global__ void gDrmsClustersMax(const int msize, const int nframes, const float* gclust_dmtx, const int cluster, const float cutoff, const int* frameapp_read, int* frameapp_write, float* distance, const float nointrasegm_corr_fact, const int nclusters, const int clustercenter) {
	
	// msize = size of each distance matrix; nframes = number of frames used for clustering;
	// gclust_dmtx = array of ALL distance matrices; cluster = number of the cluster; frameapp stores the cluster center 
	// of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays; 
	// distance stores the calculated drms because it will be needed in the post processing;
	// nclusters passes the number of already found clusters; clustercenter passes the center of the current cluster

	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x; 	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	//with maxspeed frames are assigned to the first cluster -> do not check frame again if it was already assigned
	if(frameapp_read[index] != -1) {
		frameapp_write[index] = frameapp_read[index];
		return;
	}		
	float drms=0. , di;
	int jj;
	
	//calculate the drms of cluster and index
	for (jj=0;jj<msize;jj++){
		di=(gclust_dmtx[(index+nclusters)*msize+jj]-gclust_dmtx[cluster*msize+jj]);
		drms+=di*di; 
	}			
	drms = sqrtf(drms);
    drms *= nointrasegm_corr_fact/sqrtf(msize); //Renormalize the distance properly

	if (drms<cutoff){		
		frameapp_write[index] = clustercenter;
		distance[index] = drms;
		return;
	}
	
	frameapp_write[index] = frameapp_read[index];
}

// a kernel for the lfull flag for calculation with limited memory; this is for comparing the frames of the chunk to the previously found clusters
__global__ void gDrmsClustersClosest(const int msize, const int nframes, const float* gclust_dmtx, const int cluster, const int* frameapp_read, int* frameapp_write, float* distance, const float nointrasegm_corr_fact, const int nclusters, const int clustercenter) {
	
	// msize = size of each distance matrix; nframes = number of frames used for clustering;
	// gclust_dmtx = array of ALL distance matrices; cluster = number of the cluster; frameapp stores the cluster center 
	// of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays; 
	// distance stores the calculated for comparison;
	// nclusters passes the number of already found clusters; clustercenter passes the center of the current cluster

	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}		
	float drms=0. , di;
	int jj;
	
	//calculate the drms of cluster and index
	for (jj=0;jj<msize;jj++){
		di=(gclust_dmtx[(index+nclusters)*msize+jj]-gclust_dmtx[cluster*msize+jj]);
		drms+=di*di; 
	}			
	drms = sqrtf(drms);
    drms *= nointrasegm_corr_fact/sqrtf(msize); //Renormalize the distance properly

	//at the beginning distance is set to cutoff, by always comparing the drms to the current value of distance instead of the cutoff we can reassign the frame if we find a closer cluster
	if (drms < distance[index]){		
		frameapp_write[index] = clustercenter;
		distance[index] = drms;
		return;
	}
	frameapp_write[index] = frameapp_read[index];
}

// a kernel for the maxspeed flag for calculation with limited memory; this is for comparing the frames of the chunk among themselves
__global__ void gDrmsFramesMax(const int msize, const int nframes, const float* gclust_dmtx, const int cluster, const float cutoff, const int* frameapp_read, int* frameapp_write, float* distance, const float nointrasegm_corr_fact, const int framesFinished, const int nclusters, int* newClusters, int* clusterCenters ) {
	
	// msize = size of each distance matrix; nframes = number of frames used for clustering;
	// gclust_dmtx = array of ALL distance matrices; cluster = center of a possible cluster; frameapp stores the cluster center 
	// of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays;
	// distance stores the calculated drms because it will be needed in the post processing;
	// framesFinished passes the number of processed frames in a previous chunk; nclusters passes the number of already found clusters;
	// newClusters stores the number of new found clusters in this chunk; clusterCenters stores the centers of these clusters
	

	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	//with maxspeed frames are assigned to the first cluster -> do not check frame again if it was already assigned
	if(frameapp_read[index] != -1) {
		frameapp_write[index] = frameapp_read[index];
		return;
	}
	
	//if cluster has already been assigned to another cluster then it cannot be a new cluster
	if(frameapp_read[cluster] == -1) {
		
		//spare the distance calculation if index and cluster are the same, by wordoms conventions the distance is set to -1 instead of 0 in this case
		if(cluster == index) {
			frameapp_write[index] = framesFinished + cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = -1.0;
			clusterCenters[*newClusters] = framesFinished + cluster + 1;
			(*newClusters)++;
			return;
		}		
		float drms=0. , di;
		int jj;
		
		//calculate the drms of cluster and index
		for (jj=0;jj<msize;jj++){
			di=(gclust_dmtx[(index+nclusters)*msize+jj]-gclust_dmtx[(cluster+nclusters)*msize+jj]);
			drms+=di*di; 
		}			
		drms = sqrtf(drms);
	    drms *= nointrasegm_corr_fact/sqrtf((float)msize); //Renormalize the distance properly
	
		if (drms<cutoff){		
			frameapp_write[index] = framesFinished + cluster + 1;
			distance[index] = drms;
			return;
		}
	}
	frameapp_write[index] = frameapp_read[index];
}

// a kernel for the lfull flag for calculation with limited memory; this is for comparing the frames of the chunk among themselves
__global__ void gDrmsFramesClosest(const int msize, const int nframes, const float* gclust_dmtx, const int cluster, const int* frameapp_read, int* frameapp_write, float* distance, const float nointrasegm_corr_fact, const int framesFinished, const int nclusters, int* newClusters, int* clusterCenters ) {
	
	// msize = size of each distance matrix; nframes = number of frames used for clustering;
	// gclust_dmtx = array of ALL distance matrices; cluster = center of a possible cluster; frameapp stores the cluster center 
	// of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays;
	// distance stores the calculated drms for comparison;
	// framesFinished passes the number of processed frames in a previous chunk; nclusters passes the number of already found clusters;
	// newClusters stores the number of new found clusters in this chunk; clusterCenters stores the centers of these clusters	
	
	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	//frames can only be assigned to clusters with a lower frame number
	if(cluster > index) {
		frameapp_write[index] = frameapp_read[index];
		return;
	}
	
	//if cluster has already been assigned to another cluster then it cannot be a new cluster
	if(frameapp_read[cluster] == -1) {
	
		//spare the distance calculation if index and cluster are the same, by wordoms conventions the distance is set to -1 instead of 0 in this case
		if(cluster == index) {
			frameapp_write[index] = framesFinished + cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = -1.0;
			clusterCenters[*newClusters] = framesFinished + cluster + 1;
			(*newClusters)++;
			return;
		}		
		float drms=0. , di;
		int jj;
		
		//calculate the drms of cluster and index
		for (jj=0;jj<msize;jj++){
			di=(gclust_dmtx[(index+nclusters)*msize+jj]-gclust_dmtx[(cluster+nclusters)*msize+jj]);
			drms+=di*di; 
		}			
		drms = sqrtf(drms);
	    drms *= nointrasegm_corr_fact/sqrtf((float)msize); //Renormalize the distance properly
	
		//at the beginning distance is set to cutoff, by always comparing the drms to the current value of distance instead of the cutoff we can reassign the frame if we find a closer cluster
		if (drms < distance[index]){	
			frameapp_write[index] = framesFinished + cluster + 1;
			distance[index] = drms;
			return;
		}	
	}
	frameapp_write[index] = frameapp_read[index];
}

__global__ void shiftToCenter(float* gclust_coords, const int nato, const int nframes) {
  
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	float cms[3];
  
	int ii,jj;

  // calculate the centre of mass 
	for (ii=0; ii<3; ii++)
		cms[ii] = 0.0;
  
	for (ii=0; ii<nato; ii++) 
		for (jj=0; jj<3; jj++)
			cms[jj] += gclust_coords[3*nato*index+3*ii+jj];
			
    
	for (ii=0; ii<3; ii++)
		cms[ii] /= nato;


  // shift mov_xlist and ref_xlist to centre of mass
	for (ii=0; ii<nato; ii++) 
		for (jj=0; jj<3; jj++) 
			gclust_coords[3*nato*index+3*ii+jj] -= cms[jj];

}

//A macro for handling CUDA errors
void errorHandler  (hipError_t error, int line){
  if(error != hipSuccess)
  {
    // print the CUDA error message and exit
    fprintf(stderr,"CUDA error: %s in line number %d\n", hipGetErrorString(error),line);
    exit(-1);
  }
}

// the CUDA compiler generates C++ object files, thus the main procedure has to be an extern "C" for usage in wordom
extern "C" int gClusterRmsd (struct inp_Cluster *inp_cluster,float *distance) {
	
	
	int ii;
    float cutoff = inp_cluster->threshold;
   	int nato = inp_cluster->nato;
	int totframe = inp_cluster->totframe;
	//float **gclust_coords = inp_cluster->gclust_coords;
	float *gclust_coords = inp_cluster->gclust_coords;
    int *frameapp = inp_cluster->frameapp;
    int super = inp_cluster->super;
    int step = inp_cluster->step;
    int frames = totframe/step+(totframe%step == 0 ? 0 : 1); //the number of frames that have to be analysed 

	size_t coords_size = 3*nato*sizeof(float); //memory size for coords of a single frame in one dimension
	size_t memsize= frames * coords_size; //memory size for the array of coords of all frames
	size_t cmemsize= frames * sizeof(int); //memory size for the frameapp array
	size_t dmemsize= frames * sizeof(float); //memory size for the distance array
	size_t totalmemsize = memsize + 2*cmemsize + dmemsize;
	
	//float *devPtr_gclust_coordsX;
	//float *devPtr_gclust_coordsY;
	//float *devPtr_gclust_coordsZ;
	float *devPtr_gclust_coords;
	float *devPtr_distance;
	int *devPtr_frameapp1;
	int *devPtr_frameapp2;
			
	int deviceCount; // number of devices, i.e. gpus
	int device;
	int threadsPerBlock;
	struct hipDeviceProp_t properties;		
	hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
	if (cudaResultCode != hipSuccess)
		deviceCount = 0;
		
	fprintf(stderr,"Starting GPU calculation, devicecount : %d\n", deviceCount);
		
	// machines with no GPUs can still report one emulation device 	
	for (device = 0; device < deviceCount; ++device) {
		hipGetDeviceProperties(&properties, device);
		if (properties.major != 9999) // 9999 means emulation only
			if (device==0){
				fprintf(stderr,"multiProcessorCount %d\n",properties.multiProcessorCount);
				fprintf(stderr,"maxThreadsPerMultiProcessor %d\n",properties.maxThreadsPerMultiProcessor);
				
				if(properties.major == 2)
					threadsPerBlock = 448;
				else
					threadsPerBlock = 320;
			}
	}
	
	fprintf(stderr,"threads per block: %d\n",threadsPerBlock);
	
	size_t freemem;
	size_t total;
		
	//cuda API functions always return some type of error, but if no error occured, this error is just a hipSuccess
	//errorHandler terminates program in case there was no hipSuccess reported
	errorHandler(hipMemGetInfo(&freemem, &total),__LINE__);
	
		
	fprintf(stderr,"Available memory on device: %u\n Total memory necessary on device for calculation: %u\n",freemem,totalmemsize);
		

	/*if(freemem < totalmemsize) {
		
				//DEBUG fprintf(stderr,"Available memory on device: %u\n Total memory necessary on device for calculation: %u\n .. splitting up calculation\n",freemem,totalmemsize);
		
		int framesRemaining = frames; //the number of frames that still have to be analysed
		int framesFinished = 0;
		int nclusters = 0; //the number of clusters already found
		int newClusters = 0;
		int *cluster = (int*)calloc(frames,sizeof(int)); //clustercenters of existing clusters
		int *devPtr_cluster; //stores the clustercenter of new found clusters
		int *devPtr_newClusters; //stores the number of new found clusters
		int nframes;	

		while(framesRemaining > 0) {
				
			size_t clust_coords_mem = nclusters * coords_size; //additional memory for the clusters' distance matrices
			errorHandler(hipMemGetInfo(&freemem, &total),__LINE__);
			
			//number of frames that fit into memory; 2MB of the total memory reported to freemem have to remain free, allocations fail otherwise (value found by trial and error)
			nframes = (freemem -2000000 - clust_coords_mem - sizeof(int))/(coords_size+3*sizeof(int)+sizeof(float));
			//DEBUG fprintf(stderr,"Free memory: %u, Frames remaining: %d, Frames fitting into memory: %d, Number of clusters: %d\n",freemem,framesRemaining,nframes,nclusters);
			
			//nframes is either the number of frames that fit into gpu memory, or the number of remaining frames
			if(nframes >= framesRemaining) {
				nframes = framesRemaining;
			} else {
				
				//if the number of clusters gets too high the calculation has to be stopped
				if(nclusters > nframes) {
					fprintf(stderr,"Number of clusters has exceeded number of frames that fit on GPU memory, calculation is getting too slow!\n Quitting calculation... Please choose a greater cutoff!\n");
					exit(-1);
				}
			}		
			
			//recalculate the memory sizes
			size_t memsize= nframes * coords_size;
			size_t cmemsize= nframes * sizeof(int);
			size_t dmemsize= nframes * sizeof(float);	
				
			//allocating memory on the GPU
			errorHandler(hipMalloc((void**)&devPtr_gclust_coords,memsize+clust_coords_mem),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_distance,dmemsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_frameapp1,cmemsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_frameapp2,cmemsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_cluster,cmemsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_newClusters,sizeof(int)),__LINE__);
					
			//if there were already clusters found, copy their distance matrices
			if(clust_coords_mem>0){
				//because of the overhead of a single copy instruction we prefer to copy one large data packet over lots of small ones, we use a temporary array for this
				float *clusters_coords;
				clusters_coords=(float *)malloc(clust_coords_mem);
				
				for(ii = 0; ii < nclusters; ii++)
					memcpy(clusters_coords + ii*3*nato,gclust_coords + cluster[ii]*3*nato,coords_size);
				
				errorHandler(hipMemcpy(devPtr_gclust_coords,clusters_coords,clust_coords_mem,hipMemcpyHostToDevice),__LINE__);
				free(clusters_coords);
			}		
			
			//copy distance matrices of the frames to gpu, they are copied right after the distance matrices of the clusters
			errorHandler(hipMemcpy(devPtr_gclust_coords + nclusters*3*nato, gclust_coords + (framesFinished + 1)*3*nato, memsize, hipMemcpyHostToDevice),__LINE__);	
				
			//Sets all indices to -1
			errorHandler(hipMemset((void*)devPtr_frameapp1,-1,cmemsize),__LINE__);
			
			//in order to find the closest cluster we set the distances to the cutoff for the start
			if(!inp_cluster->maxspeed){
				for(ii=framesFinished; ii<=nframes+framesFinished;ii++) {
					distance[ii] = cutoff;
				}
				errorHandler(hipMemcpy(devPtr_distance, distance+framesFinished+1, dmemsize, hipMemcpyHostToDevice),__LINE__);
			}	
				
			//set number of new clusters to 0
			errorHandler(hipMemset((void*)devPtr_newClusters,0,sizeof(int)),__LINE__);
					
			int blocks = nframes/threadsPerBlock +1; //in total we want 1 thread for each frame
						
			if(inp_cluster->maxspeed) {			
				//compare the frames of the chunk to the previously found clusters first
				for(ii = 0; ii < nclusters; ii++) {		
						
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					if((ii+1)%2) gRmsdClustersMax<<<blocks, threadsPerBlock>>>(nato, nframes, devPtr_gclust_coords, ii, cutoff, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, nclusters, cluster[ii]);
					else gRmsdClustersMax<<<blocks, threadsPerBlock>>>(nato, nframes, devPtr_gclust_coords, ii, cutoff, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, nclusters, cluster[ii]);
					
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Comparing to previous clusters %% %f\r",(double)ii/nclusters*100.0);//just a progress bar
									
				}
				
				fprintf(stderr,"\n");
				
				//then check the remaining frames against each other
				for(ii=0; ii < nframes; ii++){	
						
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					if((nclusters + ii + 1)%2) gRmsdFramesMax<<<blocks, threadsPerBlock>>>(nato, nframes, devPtr_gclust_coords,ii, cutoff, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, framesFinished, nclusters, devPtr_newClusters, devPtr_cluster);
					else gRmsdFramesMax<<<blocks, threadsPerBlock>>>(nato, nframes, devPtr_gclust_coords, ii, cutoff, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, framesFinished, nclusters, devPtr_newClusters, devPtr_cluster);
					
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Calculating Stage %% %f\r",(double)(framesFinished + ii)/frames*100.0);//just a progress bar
					
				}
				
			} else {	
				//compare the frames of the chunk to the previously found clusters first						
				for(ii = 0; ii < nclusters; ii++) {		
					
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					if((ii+1)%2) gRmsdClustersClosest<<<blocks, threadsPerBlock>>>(nato, nframes, devPtr_gclust_coords, ii, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, nclusters, cluster[ii]);
					else gRmsdClustersClosest<<<blocks, threadsPerBlock>>>(nato, nframes, devPtr_gclust_coords, ii, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, nclusters, cluster[ii]);
					
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Comparing to previous clusters %% %f\r",(double)ii/nclusters*100.0);//just a progress bar
					
				}
				
				
				fprintf(stderr,"\n");
				
				//then check the remaining frames against each other	
				for(ii=0; ii < nframes; ii++){
					
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					if((nclusters + ii + 1)%2) gRmsdFramesClosest<<<blocks, threadsPerBlock>>>(nato, nframes, devPtr_gclust_coords, ii, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, framesFinished, nclusters, devPtr_newClusters, devPtr_cluster);
					else gRmsdFramesClosest<<<blocks, threadsPerBlock>>>(nato, nframes, devPtr_gclust_coords, ii, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, framesFinished, nclusters, devPtr_newClusters, devPtr_cluster);
					
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Calculating Stage %% %f\r",(double)(framesFinished + ii)/frames*100.0);//just a progress bar
					
				}
				
			}
			
			printf("\n");	
			//DEBUG fprintf(stderr,"Copying to Host ..\n");
			
			//copy back to host, by adding framesFinished/nclusters to the pointers we make sure not to overwrite the results from previous runs
			if((nclusters + ii + 1)%2) errorHandler( hipMemcpy(frameapp+framesFinished + 1, devPtr_frameapp2, cmemsize, hipMemcpyDeviceToHost),__LINE__);
			else errorHandler( hipMemcpy(frameapp+framesFinished + 1, devPtr_frameapp1, cmemsize, hipMemcpyDeviceToHost),__LINE__);
			errorHandler( hipMemcpy(distance+framesFinished + 1, devPtr_distance, dmemsize, hipMemcpyDeviceToHost),__LINE__);
			errorHandler( hipMemcpy(&newClusters, devPtr_newClusters, sizeof(int), hipMemcpyDeviceToHost),__LINE__);
			errorHandler( hipMemcpy(cluster+nclusters, devPtr_cluster,newClusters*sizeof(int),hipMemcpyDeviceToHost),__LINE__);
									
			//update number of clusters and processed frames
			nclusters += newClusters;
			framesFinished += nframes;
			framesRemaining -= nframes;	
			
			//free all the GPU memory
			errorHandler(hipFree(devPtr_gclust_coords),__LINE__);
			errorHandler(hipFree(devPtr_distance),__LINE__);
			errorHandler(hipFree(devPtr_frameapp1),__LINE__);
			errorHandler(hipFree(devPtr_frameapp2),__LINE__);
			errorHandler(hipFree(devPtr_cluster),__LINE__);
			errorHandler(hipFree(devPtr_newClusters),__LINE__);			
		}	
		
			
		return 0;
	} else {*/
			
		//allocate gpu memory
		errorHandler(hipMalloc((void**)&devPtr_gclust_coords, memsize),__LINE__);
		//errorHandler(hipMalloc((void**)&devPtr_gclust_coordsX, memsize/3),__LINE__);
		//errorHandler(hipMalloc((void**)&devPtr_gclust_coordsY, memsize/3),__LINE__);
		//errorHandler(hipMalloc((void**)&devPtr_gclust_coordsZ, memsize/3),__LINE__);
		errorHandler(hipMalloc((void**)&devPtr_distance, dmemsize),__LINE__);
		errorHandler(hipMalloc((void**)&devPtr_frameapp1, cmemsize),__LINE__); 
		errorHandler(hipMalloc((void**)&devPtr_frameapp2, cmemsize),__LINE__); 
						
		//copy coords to gpu
		errorHandler(hipMemcpy(devPtr_gclust_coords, gclust_coords + 3*nato, frames*coords_size, hipMemcpyHostToDevice),__LINE__);
		//errorHandler(hipMemcpy(devPtr_gclust_coords, gclust_coords + coords_size, frames*coords_size, hipMemcpyHostToDevice),__LINE__);
		//errorHandler(hipMemcpy(devPtr_gclust_coords+frames*nato, gclust_coords[1] + coords_size, frames*coords_size, hipMemcpyHostToDevice),__LINE__);
		//errorHandler(hipMemcpy(devPtr_gclust_coords+2*frames*nato, gclust_coords[2] + coords_size, frames*coords_size, hipMemcpyHostToDevice),__LINE__);
		//errorHandler(hipMemcpy(devPtr_gclust_coords, gclust_coords + 3*nato, memsize, hipMemcpyHostToDevice),__LINE__);
		//errorHandler(hipMemcpy(devPtr_gclust_coordsX, gclust_coords[0] + nato, memsize/3, hipMemcpyHostToDevice),__LINE__);
		//errorHandler(hipMemcpy(devPtr_gclust_coordsY, gclust_coords[1] + nato, memsize/3, hipMemcpyHostToDevice),__LINE__);
		//errorHandler(hipMemcpy(devPtr_gclust_coordsZ, gclust_coords[2] + nato, memsize/3, hipMemcpyHostToDevice),__LINE__);
		
		//in order to find the closest cluster we set the distances to the cutoff for the start
		if(!inp_cluster->maxspeed){
			for(ii=0; ii<=frames;ii++) {
					distance[ii] = cutoff;
				}
			errorHandler(hipMemcpy(devPtr_distance, distance+1, dmemsize, hipMemcpyHostToDevice),__LINE__);
		}
			
		//Set all indices to -1
		errorHandler(hipMemset((void*)devPtr_frameapp1,-1,cmemsize),__LINE__);
				
	
		int blocks = frames/threadsPerBlock +1; //in total we want 1 thread for each frame
		
		if(super) {
			
			shiftToCenter<<<blocks, threadsPerBlock>>>(devPtr_gclust_coords, nato, frames);
			errorHandler( hipPeekAtLastError(),__LINE__);
		
					
			if(inp_cluster->maxspeed) {
				for(ii=0;ii< frames;ii++){
							
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					//if((ii+1)%2) gRmsdMax<<<blocks, threadsPerBlock>>>(nato, frames, devPtr_gclust_coordsX, devPtr_gclust_coordsY, devPtr_gclust_coordsZ, ii, cutoff, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, super);
					//else gRmsdMax<<<blocks, threadsPerBlock>>>(nato, frames, devPtr_gclust_coordsX, devPtr_gclust_coordsY, devPtr_gclust_coordsZ, ii, cutoff, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, super);
					
					if((ii+1)%2) gRmsdMaxSuper<<<blocks, threadsPerBlock>>>(nato, frames, devPtr_gclust_coords, ii, cutoff, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance);
					else gRmsdMaxSuper<<<blocks, threadsPerBlock>>>(nato, frames, devPtr_gclust_coords, ii, cutoff, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance);
							
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Stage %% %f\r",(double)ii/frames*100.0);//just a progress bar
							
					}
						
				} else {
					for(ii=0;ii< frames;ii++){
							
						//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
						//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
						//if((ii+1)%2) gRmsdClosest<<<blocks, threadsPerBlock>>>(nato, frames, devPtr_gclust_coords, ii, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance);
						//else gRmsdClosest<<<blocks, threadsPerBlock>>>(nato, frames, devPtr_gclust_coords, ii, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance);
							
						errorHandler( hipPeekAtLastError(),__LINE__);
						fprintf(stderr,"Stage %% %f\r",(double)ii/frames*100.0);//just a progress bar
							
					}
					
				}
			
		}	
			printf("\n");
				
			//DEBUG fprintf(stderr,"Copying results to Host ..\n");
			
			if((ii+1)%2) errorHandler( hipMemcpy(frameapp+1, devPtr_frameapp2, cmemsize, hipMemcpyDeviceToHost),__LINE__);
			else errorHandler( hipMemcpy(frameapp+1, devPtr_frameapp1, cmemsize, hipMemcpyDeviceToHost),__LINE__);
				
			errorHandler( hipMemcpy(distance+1, devPtr_distance, dmemsize, hipMemcpyDeviceToHost),__LINE__);
				
			//free GPU memory
			errorHandler( hipFree(devPtr_gclust_coords),__LINE__);
			//errorHandler( hipFree(devPtr_gclust_coordsX),__LINE__);
			//errorHandler( hipFree(devPtr_gclust_coordsY),__LINE__);
			//errorHandler( hipFree(devPtr_gclust_coordsZ),__LINE__);
			errorHandler( hipFree(devPtr_frameapp1),__LINE__);
			errorHandler( hipFree(devPtr_frameapp2),__LINE__);
			errorHandler( hipFree(devPtr_distance),__LINE__);		
			return 0;
		
	//}
		
}
	

// the CUDA compiler generates C++ object files, thus the main procedure has to be an extern "C" for usage in wordom
extern "C" int gClusterDrms (struct inp_Cluster *inp_cluster,float *distance)
{
    int ii;
    float cutoff = inp_cluster->threshold;
   	int msize = inp_cluster->msize;
	int totframe = inp_cluster->totframe;
	float *gclust_dmtx = inp_cluster->gclust_dmtx;
    float nointrasegm_corr_fact = 1.0;
    int *frameapp = inp_cluster->frameapp;
    int step = inp_cluster->step;
    int frames = totframe/step+(totframe%step == 0 ? 0 : 1); //the number of frames that have to be analysed 
     
    //change correction factor if correction should be applied
	if( inp_cluster->nointrasegm != 0)
		nointrasegm_corr_fact = inp_cluster->nointrasegm_corr_fact;

	size_t dmtx_size = msize*sizeof(float); //memory size for a single distance matrix
	size_t memsize= frames * dmtx_size; //memory size for the array of distance matrices
	size_t cmemsize= frames * sizeof(int); //memory size for the frameapp array
	size_t dmemsize= frames * sizeof(float); //memory size for the distance array
	size_t totalmemsize = memsize + cmemsize + dmemsize;
	
	float *devPtr_gclust_dmtx;
	float *devPtr_distance;
	int *devPtr_frameapp1;
	int *devPtr_frameapp2;
			
	int deviceCount; // number of devices, i.e. gpus
	int device;
	int threadsPerBlock;
	struct hipDeviceProp_t properties;		
	hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
	if (cudaResultCode != hipSuccess)
		deviceCount = 0;
		
	fprintf(stderr,"Starting GPU calculation, devicecount : %d\n", deviceCount);
		
	// machines with no GPUs can still report one emulation device 	
	for (device = 0; device < deviceCount; ++device) {
		hipGetDeviceProperties(&properties, device);
		if (properties.major != 9999) // 9999 means emulation only
			if (device==0){
				fprintf(stderr,"multiProcessorCount %d\n",properties.multiProcessorCount);
				fprintf(stderr,"maxThreadsPerMultiProcessor %d\n",properties.maxThreadsPerMultiProcessor);
				
				if(properties.major == 2)
					threadsPerBlock = 192;
				else
					threadsPerBlock = 256;
			}
	}
	size_t freemem;
	size_t total;
		
	//cuda API functions always return some type of error, but if no error occured, this error is just a hipSuccess
	//errorHandler terminates program in case there was no hipSuccess reported
	errorHandler(hipMemGetInfo(&freemem, &total),__LINE__);
	
	//check if there is enough gpu memory for the job and split up the calculation if not
	if(freemem < totalmemsize) {		
		//DEBUG fprintf(stderr,"Available memory on device: %u\n Total memory necessary on device for calculation: %u\n .. splitting up calculation\n",freemem,totalmemsize);
		
		int framesRemaining = frames; //the number of frames that still have to be analysed
		int framesFinished = 0;
		int nclusters = 0; //the number of clusters already found
		int newClusters = 0;
		int *cluster = (int*)calloc(frames,sizeof(int)); //clustercenters of existing clusters
		int *devPtr_cluster; //stores the clustercenter of new found clusters
		int *devPtr_newClusters; //stores the number of new found clusters
		int nframes;	

		while(framesRemaining > 0) {
				
			size_t clust_dmtx_mem = nclusters * dmtx_size; //additional memory for the clusters' distance matrices
			errorHandler(hipMemGetInfo(&freemem, &total),__LINE__);
			
			//number of frames that fit into memory; 2MB of the total memory reported to freemem have to remain free, allocations fail otherwise (value found by trial and error)
			nframes = (freemem -2000000 - clust_dmtx_mem - sizeof(int))/(dmtx_size+3*sizeof(int)+sizeof(float));
			//DEBUG fprintf(stderr,"Free memory: %u, Frames remaining: %d, Frames fitting into memory: %d, Number of clusters: %d\n",freemem,framesRemaining,nframes,nclusters);
			
			//nframes is either the number of frames that fit into gpu memory, or the number of remaining frames
			if(nframes >= framesRemaining) {
				nframes = framesRemaining;
			} else {
				
				//if the number of clusters gets too high the calculation has to be stopped
				if(nclusters > nframes) {
					fprintf(stderr,"Number of clusters has exceeded number of frames that fit on GPU memory, calculation is getting too slow!\n Quitting calculation... Please choose a greater cutoff!\n");
					exit(-1);
				}
			}		
			
			//recalculate the memory sizes
			size_t memsize= nframes * dmtx_size;
			size_t cmemsize= nframes * sizeof(int);
			size_t dmemsize= nframes * sizeof(float);	
				
			//allocating memory on the GPU
			errorHandler(hipMalloc((void**)&devPtr_gclust_dmtx,memsize+clust_dmtx_mem),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_distance,dmemsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_frameapp1,cmemsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_frameapp2,cmemsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_cluster,cmemsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_newClusters,sizeof(int)),__LINE__);
					
			//if there were already clusters found, copy their distance matrices
			if(clust_dmtx_mem>0){
				//because of the overhead of a single copy instruction we prefer to copy one large data packet over lots of small ones, we use a temporary array for this
				float *clusters_dmtx;
				clusters_dmtx=(float *)malloc(clust_dmtx_mem);
				
				for(ii = 0; ii < nclusters; ii++)
					memcpy(clusters_dmtx + ii*msize,gclust_dmtx + cluster[ii]*msize,dmtx_size);
				
				errorHandler(hipMemcpy(devPtr_gclust_dmtx,clusters_dmtx,clust_dmtx_mem,hipMemcpyHostToDevice),__LINE__);
				free(clusters_dmtx);
			}		
			
			//copy distance matrices of the frames to gpu, they are copied right after the distance matrices of the clusters
			errorHandler(hipMemcpy(devPtr_gclust_dmtx + nclusters*msize, gclust_dmtx + (framesFinished + 1)*msize, memsize, hipMemcpyHostToDevice),__LINE__);	
				
			//Sets all indices to -1
			errorHandler(hipMemset((void*)devPtr_frameapp1,-1,cmemsize),__LINE__);
			
			//in order to find the closest cluster we set the distances to the cutoff for the start
			if(!inp_cluster->maxspeed){
				for(ii=framesFinished; ii<=nframes+framesFinished;ii++) {
					distance[ii] = cutoff;
				}
				errorHandler(hipMemcpy(devPtr_distance, distance+framesFinished+1, dmemsize, hipMemcpyHostToDevice),__LINE__);
			}	
				
			//set number of new clusters to 0
			errorHandler(hipMemset((void*)devPtr_newClusters,0,sizeof(int)),__LINE__);
					
			int blocks = nframes/threadsPerBlock +1; //in total we want 1 thread for each frame
						
			if(inp_cluster->maxspeed) {			
				//compare the frames of the chunk to the previously found clusters first
				for(ii = 0; ii < nclusters; ii++) {		
							
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					if((ii+1)%2) gDrmsClustersMax<<<blocks, threadsPerBlock>>>(msize, nframes, devPtr_gclust_dmtx, ii, cutoff, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, nointrasegm_corr_fact, nclusters, cluster[ii]);
					else gDrmsClustersMax<<<blocks, threadsPerBlock>>>(msize, nframes, devPtr_gclust_dmtx, ii, cutoff, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, nointrasegm_corr_fact, nclusters, cluster[ii]);
					
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Comparing to previous clusters %% %f\r",(double)ii/nclusters*100.0);//just a progress bar
									
				}
				
				fprintf(stderr,"\n");
				
				//then check the remaining frames against each other
				for(ii=0; ii < nframes; ii++){	
						
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					if((nclusters + ii + 1)%2) gDrmsFramesMax<<<blocks, threadsPerBlock>>>(msize, nframes, devPtr_gclust_dmtx,ii, cutoff, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, nointrasegm_corr_fact, framesFinished, nclusters, devPtr_newClusters, devPtr_cluster);
					else gDrmsFramesMax<<<blocks, threadsPerBlock>>>(msize, nframes, devPtr_gclust_dmtx, ii, cutoff, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, nointrasegm_corr_fact, framesFinished, nclusters, devPtr_newClusters, devPtr_cluster);
					
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Calculating Stage %% %f\r",(double)(framesFinished + ii)/frames*100.0);//just a progress bar
					
				}
				
			} else {	
				//compare the frames of the chunk to the previously found clusters first						
				for(ii = 0; ii < nclusters; ii++) {		
					
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					if((ii+1)%2) gDrmsClustersClosest<<<blocks, threadsPerBlock>>>(msize, nframes, devPtr_gclust_dmtx, ii, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, nointrasegm_corr_fact, nclusters, cluster[ii]);
					else gDrmsClustersClosest<<<blocks, threadsPerBlock>>>(msize, nframes, devPtr_gclust_dmtx, ii, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, nointrasegm_corr_fact, nclusters, cluster[ii]);
					
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Comparing to previous clusters %% %f\r",(double)ii/nclusters*100.0);//just a progress bar
					
				}
				
				fprintf(stderr,"\n");
				
				//then check the remaining frames against each other	
				for(ii=0; ii < nframes; ii++){
					
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					if((nclusters + ii + 1)%2) gDrmsFramesClosest<<<blocks, threadsPerBlock>>>(msize, nframes, devPtr_gclust_dmtx, ii, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, nointrasegm_corr_fact, framesFinished, nclusters, devPtr_newClusters, devPtr_cluster);
					else gDrmsFramesClosest<<<blocks, threadsPerBlock>>>(msize, nframes, devPtr_gclust_dmtx, ii, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, nointrasegm_corr_fact, framesFinished, nclusters, devPtr_newClusters, devPtr_cluster);
					
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Calculating Stage %% %f\r",(double)(framesFinished + ii)/frames*100.0);//just a progress bar
					
				}
			}
			
			printf("\n");	
			//DEBUG fprintf(stderr,"Copying to Host ..\n");
			
			//copy back to host, by adding framesFinished/nclusters to the pointers we make sure not to overwrite the results from previous runs
			if((nclusters + ii + 1)%2) errorHandler( hipMemcpy(frameapp+framesFinished + 1, devPtr_frameapp2, cmemsize, hipMemcpyDeviceToHost),__LINE__);
			else errorHandler( hipMemcpy(frameapp+framesFinished + 1, devPtr_frameapp1, cmemsize, hipMemcpyDeviceToHost),__LINE__);
			errorHandler( hipMemcpy(distance+framesFinished + 1, devPtr_distance, dmemsize, hipMemcpyDeviceToHost),__LINE__);
			errorHandler( hipMemcpy(&newClusters, devPtr_newClusters, sizeof(int), hipMemcpyDeviceToHost),__LINE__);
			errorHandler( hipMemcpy(cluster+nclusters, devPtr_cluster,newClusters*sizeof(int),hipMemcpyDeviceToHost),__LINE__);
									
			//update number of clusters and processed frames
			nclusters += newClusters;
			framesFinished += nframes;
			framesRemaining -= nframes;	
			
			//free all the GPU memory
			errorHandler(hipFree(devPtr_gclust_dmtx),__LINE__);
			errorHandler(hipFree(devPtr_distance),__LINE__);
			errorHandler(hipFree(devPtr_frameapp1),__LINE__);
			errorHandler(hipFree(devPtr_frameapp2),__LINE__);
			errorHandler(hipFree(devPtr_cluster),__LINE__);
			errorHandler(hipFree(devPtr_newClusters),__LINE__);			
		}	
		
			
		return 0;		
		
		} else {
			
			//allocate gpu memory
			errorHandler(hipMalloc((void**)&devPtr_gclust_dmtx, memsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_distance, dmemsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_frameapp1, cmemsize),__LINE__); 
			errorHandler(hipMalloc((void**)&devPtr_frameapp2, cmemsize),__LINE__); 
					
			//copy distance matrices to gpu
			errorHandler(hipMemcpy(devPtr_gclust_dmtx, gclust_dmtx + msize, memsize, hipMemcpyHostToDevice),__LINE__);
		
			//Set all indices to -1
			errorHandler(hipMemset((void*)devPtr_frameapp1,-1,cmemsize),__LINE__);
			
			//in order to find the closest cluster we set the distances to the cutoff for the start
			if(!inp_cluster->maxspeed){
				for(ii=0; ii<=frames;ii++) {
					distance[ii] = cutoff;
				}
				errorHandler(hipMemcpy(devPtr_distance, distance+1, dmemsize, hipMemcpyHostToDevice),__LINE__);
			}

			int blocks = frames/threadsPerBlock +1; //in total we want 1 thread for each frame
				
			if(inp_cluster->maxspeed) {
				for(ii=0;ii< frames;ii++){
					
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					if((ii+1)%2) gDrmsMax<<<blocks, threadsPerBlock>>>(msize, frames, devPtr_gclust_dmtx,ii, cutoff, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, nointrasegm_corr_fact);
					else gDrmsMax<<<blocks, threadsPerBlock>>>(msize, frames, devPtr_gclust_dmtx,ii, cutoff, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, nointrasegm_corr_fact);
					
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Stage %% %f\r",(double)ii/frames*100.0);//just a progress bar
					
				}
				
			} else {
				for(ii=0;ii< frames;ii++){
					
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					if((ii+1)%2) gDrmsClosest<<<blocks, threadsPerBlock>>>(msize, frames, devPtr_gclust_dmtx, ii, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, nointrasegm_corr_fact);
					else gDrmsClosest<<<blocks, threadsPerBlock>>>(msize, frames, devPtr_gclust_dmtx, ii, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, nointrasegm_corr_fact);
					
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Stage %% %f\r",(double)ii/frames*100.0);//just a progress bar
					
				}	
			}	
			printf("\n");
			
			//DEBUG fprintf(stderr,"Copying results to Host ..\n");
		
			if((ii+1)%2) errorHandler( hipMemcpy(frameapp+1, devPtr_frameapp2, cmemsize, hipMemcpyDeviceToHost),__LINE__);
			else errorHandler( hipMemcpy(frameapp+1, devPtr_frameapp1, cmemsize, hipMemcpyDeviceToHost),__LINE__);
			
			errorHandler( hipMemcpy(distance+1, devPtr_distance, dmemsize, hipMemcpyDeviceToHost),__LINE__);
			
			//free GPU memory
			errorHandler( hipFree(devPtr_gclust_dmtx),__LINE__);
			errorHandler( hipFree(devPtr_frameapp1),__LINE__);
			errorHandler( hipFree(devPtr_frameapp2),__LINE__);
			errorHandler( hipFree(devPtr_distance),__LINE__);		
			return 0;
	   }
	}
