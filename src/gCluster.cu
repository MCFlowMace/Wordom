#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
//
// This program is free software; you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation; either version 2 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program; if not, write to the Free Software
// Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307  USA
// ------------------------------------------------------------------


#include <math.h>
#include <stdio.h> 
#include <stdlib.h>

#define GCLUSTER_INCLUDES

#include "fileio.h"
#include "cluster.h"

__device__ void normalize(float a[3])
{
  float  b;

  b = sqrtf((float)(a[0]*a[0] + a[1]*a[1] + a[2]*a[2]));
  a[0] /= b;
  a[1] /= b;
  a[2] /= b;
}



__device__ float dot(float a[3], float b[3])
{
  return (a[0] * b[0] + a[1] * b[1] + a[2] * b[2]);
}



__device__ static void cross(float a[3], float b[3], float c[3])
{
  a[0] = b[1]*c[2] - b[2]*c[1];
  a[1] = b[2]*c[0] - b[0]*c[2];
  a[2] = b[0]*c[1] - b[1]*c[0];
}



/*
 * setup_rotation() 
 *
 * given two lists of x,y,z coordinates, constructs
 * the correlation R matrix and the E value needed to calculate the
 * least-squares rotation matrix.
 */
__device__ void setup_rotation(const float *ref_xlist, const float *mov_xlist, int& n_list, float R[3][3], float& E0, volatile int& i, volatile int&j, volatile int& n)
{
  // initialize
  for (i=0; i<3; i++)
    for (j=0; j<3; j++) 
      R[i][j] = 0.0f;
  E0 = 0.0f;

  for (n=0; n<n_list; n++) 
  {
    /* 
     * E0 = 1/2 * sum(over n): y(n)*y(n) + x(n)*x(n) 
     */
    for (i=0; i<3; i++)
      E0 +=  mov_xlist[3*n+i] * mov_xlist[3*n+i]  
            + ref_xlist[3*n+i] * ref_xlist[3*n+i];
    
    /*
     * correlation matrix R:   
     *   R[i,j) = sum(over n): y(n,i) * x(n,j)  
     *   where x(n) and y(n) are two vector sets   
     */
    for (i=0; i<3; i++)
    {
      for (j=0; j<3; j++)
        R[i][j] += mov_xlist[3*n+i] * ref_xlist[3*n+j];
    }
  }
  E0 *= 0.5f;
  }



#define ROTATE(a,i,j,k,l) { g = a[i][j]; \
                            h = a[k][l]; \
                            a[i][j] = g-s*(h+g*tau); \
                            a[k][l] = h+s*(g-h*tau); }
/*   
 * jacobi3
 *
 * computes eigenval and eigen_vec of a real 3x3
 * symmetric matrix. On output, elements of a that are above 
 * the diagonal are destroyed. d[1..3] returns the 
 * eigenval of a. v[1..3][1..3] is a matrix whose 
 * columns contain, on output, the normalized eigen_vec of
 * a. n_rot returns the number of Jacobi rotations that were required.
 */
__device__ int jacobi3(float a[3][3], float d[3], float v[3][3], volatile int& k, volatile int& i, volatile int& j)
{
  float b[3], z[3];

  //Initialize v to the identity matrix.
  for (i=0; i<3; i++) 
  { 
    for (j=0; j<3; j++) 
      v[i][j] = 0.0f;
    v[i][i] = 1.0f;
  }

  // Initialize b and d to the diagonal of a
  for (i=0; i<3; i++) 
    b[i] = d[i] = a[i][i];

  // z will accumulate terms
  for (i=0; i<3; i++) 
    z[i] = 0.0f; 

  // 50 tries
  int count;
  for (count=0; count<50; count++)     
  {	
	float tresh;
    // sum off-diagonal elements
    {
	    float sum = 0.0f;
	    for (i=0; i<2; i++) 
	      for (j=i+1; j<3; j++)
	         sum += fabsf(a[i][j]);
	
	    // if converged to machine underflow
	    if (sum == 0.0f) 
	      return(1);
	
	    // on 1st three sweeps...
	    if (count < 3) 
	      tresh = sum * 0.2f / 9.0f;    
	    else       
	      tresh = 0.0f;   
	}

    for (i=0; i<2; i++) 
    {
      for (j=i+1; j<3; j++) 
      {
        float g = 100.0f * fabsf(a[i][j]);

        /*  after four sweeps, skip the rotation if
         *   the off-diagonal element is small 
         */
       if ( count > 3  &&  fabsf(d[i])+g == fabsf(d[i]) &&  fabsf(d[j])+g == fabsf(d[j]) ) 
        {
          a[i][j] = 0.0f;
        } 
        else if (fabsf(a[i][j]) > tresh) 
        {
          float h = d[j] - d[i];
          float t;
          
          if (fabsf(h)+g == fabsf(h))
          {
            t = a[i][j] / h;
          }
          else 
          {
            float theta = 0.5f * h / (a[i][j]);
            t = 1.0f / ( fabsf(theta) +
                        (float)sqrtf(1.0f + theta*theta) );
            if (theta < 0.0f) 
              t = -t;
          }
          
          float c = 1.0f / (float) sqrtf(1.0f + t*t);
          float s = t * c;
          float tau = s / (1.0f + c);
          h = t * a[i][j];

          z[i] -= h;
          z[j] += h;
          d[i] -= h;
          d[j] += h;

          a[i][j] = 0.0f;

          for (k=0; k<=i-1; k++) 
            ROTATE(a, k, i, k, j)

          for (k=i+1; k<=j-1; k++) 
            ROTATE(a, i, k, k, j)

          for (k=j+1; k<3; k++) 
            ROTATE(a, i, k, j, k)

          for (k=0; k<3; k++) 
            ROTATE(v, k, i, k, j)

        }
      }
    }

    for (i=0; i<3; i++) 
    {
      b[i] += z[i];
      d[i] = b[i];
      z[i] = 0.0f;
    }
  }

  printf("Too many iterations in jacobi3\n");
  return (0);
}  



/* 
 * diagonalize_symmetric 
 *
 *    Diagonalize a 3x3 matrix & sort eigenval by size
 */
__device__ int diagonalize_symmetric(float matrix[3][3], float vec[3][3], float eigenval[3], volatile int& i, volatile int& j, volatile int& k)
{
  
	if (!jacobi3(matrix, eigenval, vec, i,j,k))
		return (0);
	
	// sort solutions by eigenval
	for (i=0; i<3; i++) 
	{
		k = i;
		matrix[0][0] = eigenval[i];
		
		for (j=i+1; j<3; j++)
			if (eigenval[j] >= matrix[0][0])
			{ 
			k = j;
			matrix[0][0] = eigenval[k];
			}
		   
		if (k != i) 
		{
			eigenval[k] = eigenval[i];
			eigenval[i] = matrix[0][0];
			for (j=0; j<3; j++) 
			{
				matrix[0][0] = vec[j][i];
				vec[j][i] = vec[j][k];
				vec[j][k] = matrix[0][0];
			}
		}
	}
	
	// transpose such that first index refers to solution index
	for (i=0; i<3; i++)
		for (j=0; j<3; j++)
			vec[i][j] = vec[j][i];
	
	return (1);
}



/*
 * calculate_rotation() 
 *
 * calculates the residual from the R matrix and E0:
 * to reduce the number of used variables E0 has to be passed as the value of residual
 */

__device__ int calculate_rotation(float R[3][3], float& residual, volatile int& i, volatile int& j, volatile int& k)
{
	// build Rt, transpose of R 
	float Rt[3][3];
	for (i=0; i<3; i++)
		for (j=0; j<3; j++)
			Rt[i][j] = R[j][i];

	// make symmetric R = Rt X R
	float right_eigenvec[3][3];
		for (i=0; i<3; i++) 
			for (j=0; j<3; j++)
			{
				right_eigenvec[i][j] = 0.0f;
				for (k = 0; k<3; k++)
					right_eigenvec[i][j] += Rt[k][i] * R[j][k];
		    }
    
	for (i=0; i<3; i++) 
		for (j=0; j<3; j++)
			R[i][j] = right_eigenvec[i][j];

	float eigenval[3];
	if (!diagonalize_symmetric(R, right_eigenvec, eigenval,i,j,k))
	    return(0);
  

	/* right_eigenvec's should be an orthogonal system but could be left
	* or right-handed. Let's force into right-handed system.
	*/
	cross(&right_eigenvec[2][0], &right_eigenvec[0][0], &right_eigenvec[1][0]);
	
	/* From the Kabsch algorithm, the eigenvec's of RtR
	* are identical to the right_eigenvec's of R.
	* This means that left_eigenvec = R x right_eigenvec 
	*/	

	for (i=0; i<3; i++) 
		for (j=0; j<3; j++) 
			R[i][j] = dot(&right_eigenvec[i][0], &Rt[j][0]);
      	
	for (i=0; i<3; i++) 
		normalize(&R[i][0]);

	/* 
	* Force left_eigenvec[2] to be orthogonal to the other vectors.
	* First check if the rotational matrices generated from the 
	* orthogonal eigenvectors are in a right-handed or left-handed
	* co-ordinate system - given by sigma. Sigma is needed to
	* resolve this ambiguity in calculating the RMSD.
	*/

	cross(&right_eigenvec[0][0], &R[0][0], &R[1][0]);
	
	float sigma;
	if (dot(&right_eigenvec[0][0], &R[2][0]) < 0.0)
		sigma = -1.0f;
	
	else 
		sigma = 1.0f;
	
	residual = residual - (float) sqrtf(fabsf(eigenval[0])) - (float) sqrtf(fabsf(eigenval[1]))- sigma * (float) sqrtf(fabsf(eigenval[2]));	
	return (1);
}

__global__ void rmsd_kernel(int nato, const int nframes, const float* gclust_coords, const int cluster, const int* frameapp_read, int* frameapp_write, float* distance, const int mode) {
	
	/* nato = number of atoms, nframes = number of frames used for clustering;
	 * gclust_coords = array of all coordinates of the frames; cluster = center of a possible cluster; frameapp stores the cluster center
	 * of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays;
	 * distance stores the calculated rmsd because it will be needed in the post processing
	 * mode is used to switch between maxspeed=1 and closest=0
	 */
	
	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	if(mode) {
		//with the maxspeed flag frames are assigned to the first cluster -> do not check frame again if it was already assigned
		if(frameapp_read[index] != -1) {
			frameapp_write[index] = frameapp_read[index];
			return;
		}
	} else {
		//frames can only be assigned to clusters with a lower frame number
		if(cluster > index) {
			frameapp_write[index] = frameapp_read[index];
			return;
		}	
	}
	
	//if cluster has already been assigned to another cluster then it cannot be a new cluster
	if(frameapp_read[cluster] == -1) {

		//spare the distance calculation if index and cluster are the same, by wordoms conventions the distance is set to -1 instead of 0 in this case
		if(cluster == index) {
			frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = -1.0f;
			return;
		}
		
		int ii;
		float rmsd,di;
  
		rmsd=0.0f;
		di=0.0f;
  
		for ( ii=0; ii<3*nato; ii++ ) {
			di= gclust_coords[cluster*3*nato+ii]-gclust_coords[index*3*nato+ii];
			rmsd += di*di;            
		}
  
		rmsd /= nato;
		rmsd = sqrtf ( rmsd );
	
		if (rmsd<distance[index]){
			frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = rmsd;
			return;
		}	
	}
	
	frameapp_write[index] = frameapp_read[index];
}

__global__ void rmsd_super_kernel(int nato, const int nframes, const float* gclust_coords, const int cluster, const int* frameapp_read, int* frameapp_write, float* distance, const int mode) {
	
	
	/* nato = number of atoms, nframes = number of frames used for clustering;
	 * gclust_coords = array of all coordinates of the frames; cluster = center of a possible cluster; frameapp stores the cluster center 
	 * of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays;
	 * distance stores the calculated rmsd because it will be needed in the post processing
	 * mode is used to switch between maxspeed=1 and closest=0 
	 */
	

	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	if(mode) {
		//with the maxspeed flag frames are assigned to the first cluster -> do not check frame again if it was already assigned
		if(frameapp_read[index] != -1) {
			frameapp_write[index] = frameapp_read[index];
			return;
		}
	} else {
		//frames can only be assigned to clusters with a lower frame number
		if(cluster > index) {
			frameapp_write[index] = frameapp_read[index];
			return;
		}	
	}
	
	//if cluster has already been assigned to another cluster then it cannot be a new cluster
	if(frameapp_read[cluster] == -1) {

		//spare the distance calculation if index and cluster are the same, by wordoms conventions the distance is set to -1 instead of 0 in this case
		if(cluster == index) {
			frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = -1.0f;
			return;
		}
		
		float rmsd;
		float R[3][3];
		volatile int i,j,n;
		
		setup_rotation(&gclust_coords[cluster*3*nato],&gclust_coords[index*3*nato], nato, R, rmsd, i, j, n);
		
		if(calculate_rotation(R, rmsd, i, j, n)) {
  
			rmsd = fabsf(rmsd); // avoids the awkward case of -0.0 
			rmsd = sqrtf( fabsf((float) (rmsd)*2.0f/((float)nato)) );
			
			if (rmsd<distance[index]){
				frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
				distance[index] = rmsd;
				return;
			}
		}		
	}
	
	frameapp_write[index] = frameapp_read[index];			
}

// a kernel for the maxspeed flag
__global__ void gDrmsMax(const int msize, const int nframes, const float* gclust_dmtx, const int cluster, const float cutoff, const int* frameapp_read, int* frameapp_write, float* distance, const float nointrasegm_corr_fact) {
	
	/* msize = size of each distance matrix, nframes = number of frames used for clustering;
	 * gclust_dmtx = array of ALL distance matrices; cluster = center of a possible cluster; frameapp stores the cluster center
	 * of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays;
	 * distance stores the calculated drms because it will be needed in the post processing 
	 */
	
	

	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	//with the maxspeed flag frames are assigned to the first cluster -> do not check frame again if it was already assigned
	if(frameapp_read[index] != -1) {
		frameapp_write[index] = frameapp_read[index];
		return;
	}
	
	//if cluster has already been assigned to another cluster then it cannot be a new cluster
	if(frameapp_read[cluster] == -1) {
		
		//spare the distance calculation if index and cluster are the same, by wordoms conventions the distance is set to -1 instead of 0 in this case
		if(cluster == index) {
			frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = -1.0;
			return;
		}
		
		
		float drms=0. , di;
		int jj;
		
		//calculate the drms of cluster and index
		for (jj=0;jj<msize;jj++){
			di=(gclust_dmtx[index*msize+jj]-gclust_dmtx[cluster*msize+jj]);
			drms+=di*di; 
		}		
		drms = sqrtf(drms);
	    drms *= nointrasegm_corr_fact/sqrtf((float)msize); //Renormalize the distance properly
	
		if (drms<cutoff){		
			frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = drms;
			return;
		}
		
	}
	frameapp_write[index] = frameapp_read[index];
}

// a kernel for the lfull flag
__global__ void gDrmsClosest(const int msize, const int nframes, const float* gclust_dmtx, const int cluster, const int* frameapp_read, int* frameapp_write, float* distance, const float nointrasegm_corr_fact) {
	
	/* msize = size of each distance matrix; nframes = number of frames used for clustering;
	 * gclust_dmtx = array of ALL distance matrices; cluster = center of a possible cluster; frameapp stores the cluster center
	 * of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays;
	 * distance stores the calculated drms for comparison 
	 */

	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	//frames can only be assigned to clusters with a lower frame number
	if(cluster > index) {
		frameapp_write[index] = frameapp_read[index];
		return;
	}
	
	//if cluster has already been assigned to another cluster then it cannot be a new cluster
	if(frameapp_read[cluster] == -1) {
	
		//spare the distance calculation if index and cluster are the same, by wordoms conventions the distance is set to -1 instead of 0 in this case
		if(cluster == index) {
			frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = -1.0;
			return;
		}		
		float drms=0. , di;
		int jj;
		
		//calculate the drms of cluster and index
		for (jj=0;jj<msize;jj++){
			di=(gclust_dmtx[index*msize+jj]-gclust_dmtx[cluster*msize+jj]);
			drms+=di*di; 
		}			
		drms = sqrtf(drms);
	    drms *= nointrasegm_corr_fact/sqrtf((float)msize); //Renormalize the distance properly
	
		//at the beginning distance is set to the cutoff, by always comparing the drms to the current value of distance instead of only the cutoff we can reassign the frame if we find a closer cluster
		if (drms < distance[index]){		
			frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = drms;
			return;
		}
	}
	
	frameapp_write[index] = frameapp_read[index];
}

// a kernel for the maxspeed flag for calculation with limited memory; this is for comparing the frames of the chunk to the previously found clusters
__global__ void gDrmsClustersMax(const int msize, const int nframes, const float* gclust_dmtx, const int cluster, const float cutoff, const int* frameapp_read, int* frameapp_write, float* distance, const float nointrasegm_corr_fact, const int nclusters, const int clustercenter) {
	
	/* msize = size of each distance matrix; nframes = number of frames used for clustering;
	 * gclust_dmtx = array of ALL distance matrices; cluster = number of the cluster; frameapp stores the cluster center 
	 * of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays; 
	 * distance stores the calculated drms because it will be needed in the post processing;
	 * nclusters passes the number of already found clusters; clustercenter passes the center of the current cluster 
	 */

	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x; 	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	//with maxspeed frames are assigned to the first cluster -> do not check frame again if it was already assigned
	if(frameapp_read[index] != -1) {
		frameapp_write[index] = frameapp_read[index];
		return;
	}		
	float drms=0. , di;
	int jj;
	
	//calculate the drms of cluster and index
	for (jj=0;jj<msize;jj++){
		di=(gclust_dmtx[(index+nclusters)*msize+jj]-gclust_dmtx[cluster*msize+jj]);
		drms+=di*di; 
	}			
	drms = sqrtf(drms);
    drms *= nointrasegm_corr_fact/sqrtf(msize); //Renormalize the distance properly

	if (drms<cutoff){		
		frameapp_write[index] = clustercenter;
		distance[index] = drms;
		return;
	}
	
	frameapp_write[index] = frameapp_read[index];
}

// a kernel for the lfull flag for calculation with limited memory; this is for comparing the frames of the chunk to the previously found clusters
__global__ void gDrmsClustersClosest(const int msize, const int nframes, const float* gclust_dmtx, const int cluster, const int* frameapp_read, int* frameapp_write, float* distance, const float nointrasegm_corr_fact, const int nclusters, const int clustercenter) {
	
	/* msize = size of each distance matrix; nframes = number of frames used for clustering;
	 * gclust_dmtx = array of ALL distance matrices; cluster = number of the cluster; frameapp stores the cluster center 
	 * of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays; 
	 * distance stores the calculated for comparison;
	 * nclusters passes the number of already found clusters; clustercenter passes the center of the current cluster 
	 */

	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}		
	float drms=0. , di;
	int jj;
	
	//calculate the drms of cluster and index
	for (jj=0;jj<msize;jj++){
		di=(gclust_dmtx[(index+nclusters)*msize+jj]-gclust_dmtx[cluster*msize+jj]);
		drms+=di*di; 
	}			
	drms = sqrtf(drms);
    drms *= nointrasegm_corr_fact/sqrtf(msize); //Renormalize the distance properly

	//at the beginning distance is set to cutoff, by always comparing the drms to the current value of distance instead of the cutoff we can reassign the frame if we find a closer cluster
	if (drms < distance[index]){		
		frameapp_write[index] = clustercenter;
		distance[index] = drms;
		return;
	}
	frameapp_write[index] = frameapp_read[index];
}

// a kernel for the maxspeed flag for calculation with limited memory; this is for comparing the frames of the chunk among themselves
__global__ void gDrmsFramesMax(const int msize, const int nframes, const float* gclust_dmtx, const int cluster, const float cutoff, const int* frameapp_read, int* frameapp_write, float* distance, const float nointrasegm_corr_fact, const int framesFinished, const int nclusters, int* newClusters, int* clusterCenters ) {
	
	/* msize = size of each distance matrix; nframes = number of frames used for clustering;
	 * gclust_dmtx = array of ALL distance matrices; cluster = center of a possible cluster; frameapp stores the cluster center 
	 * of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays;
	 * distance stores the calculated drms because it will be needed in the post processing;
	 * framesFinished passes the number of processed frames in a previous chunk; nclusters passes the number of already found clusters;
	 * newClusters stores the number of new found clusters in this chunk; clusterCenters stores the centers of these clusters 
	 */
	

	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	//with maxspeed frames are assigned to the first cluster -> do not check frame again if it was already assigned
	if(frameapp_read[index] != -1) {
		frameapp_write[index] = frameapp_read[index];
		return;
	}
	
	//if cluster has already been assigned to another cluster then it cannot be a new cluster
	if(frameapp_read[cluster] == -1) {
		
		//spare the distance calculation if index and cluster are the same, by wordoms conventions the distance is set to -1 instead of 0 in this case
		if(cluster == index) {
			frameapp_write[index] = framesFinished + cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = -1.0;
			clusterCenters[*newClusters] = framesFinished + cluster + 1;
			(*newClusters)++;
			return;
		}		
		float drms=0. , di;
		int jj;
		
		//calculate the drms of cluster and index
		for (jj=0;jj<msize;jj++){
			di=(gclust_dmtx[(index+nclusters)*msize+jj]-gclust_dmtx[(cluster+nclusters)*msize+jj]);
			drms+=di*di; 
		}			
		drms = sqrtf(drms);
	    drms *= nointrasegm_corr_fact/sqrtf((float)msize); //Renormalize the distance properly
	
		if (drms<cutoff){		
			frameapp_write[index] = framesFinished + cluster + 1;
			distance[index] = drms;
			return;
		}
	}
	frameapp_write[index] = frameapp_read[index];
}

// a kernel for the lfull flag for calculation with limited memory; this is for comparing the frames of the chunk among themselves
__global__ void gDrmsFramesClosest(const int msize, const int nframes, const float* gclust_dmtx, const int cluster, const int* frameapp_read, int* frameapp_write, float* distance, const float nointrasegm_corr_fact, const int framesFinished, const int nclusters, int* newClusters, int* clusterCenters ) {
	
	/* msize = size of each distance matrix; nframes = number of frames used for clustering;
	 * gclust_dmtx = array of ALL distance matrices; cluster = center of a possible cluster; frameapp stores the cluster center 
	 * of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays;
	 * distance stores the calculated drms for comparison;
	 * framesFinished passes the number of processed frames in a previous chunk; nclusters passes the number of already found clusters;
	 * newClusters stores the number of new found clusters in this chunk; clusterCenters stores the centers of these clusters
	 */
	
	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	//frames can only be assigned to clusters with a lower frame number
	if(cluster > index) {
		frameapp_write[index] = frameapp_read[index];
		return;
	}
	
	//if cluster has already been assigned to another cluster then it cannot be a new cluster
	if(frameapp_read[cluster] == -1) {
	
		//spare the distance calculation if index and cluster are the same, by wordoms conventions the distance is set to -1 instead of 0 in this case
		if(cluster == index) {
			frameapp_write[index] = framesFinished + cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = -1.0;
			clusterCenters[*newClusters] = framesFinished + cluster + 1;
			(*newClusters)++;
			return;
		}		
		float drms=0. , di;
		int jj;
		
		//calculate the drms of cluster and index
		for (jj=0;jj<msize;jj++){
			di=(gclust_dmtx[(index+nclusters)*msize+jj]-gclust_dmtx[(cluster+nclusters)*msize+jj]);
			drms+=di*di; 
		}			
		drms = sqrtf(drms);
	    drms *= nointrasegm_corr_fact/sqrtf((float)msize); //Renormalize the distance properly
	
		//at the beginning distance is set to cutoff, by always comparing the drms to the current value of distance instead of the cutoff we can reassign the frame if we find a closer cluster
		if (drms < distance[index]){	
			frameapp_write[index] = framesFinished + cluster + 1;
			distance[index] = drms;
			return;
		}	
	}
	frameapp_write[index] = frameapp_read[index];
}

//shifts the center of mass to the origin
__global__ void shiftToCenter(float* gclust_coords, const int nato, const int nframes) {
  
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	float cms[3];
  
	int ii,jj;

	// calculate the centre of mass 
	for (ii=0; ii<3; ii++)
		cms[ii] = 0.0;
  
	for (ii=0; ii<nato; ii++) 
		for (jj=0; jj<3; jj++)
			cms[jj] += gclust_coords[3*nato*index+3*ii+jj];
			
    
	for (ii=0; ii<3; ii++)
		cms[ii] /= nato;


	// shift mov_xlist and ref_xlist to centre of mass
	for (ii=0; ii<nato; ii++) 
		for (jj=0; jj<3; jj++) 
			gclust_coords[3*nato*index+3*ii+jj] -= cms[jj];

}

//wrapper function for handling CUDA errors
void errorHandler  (hipError_t error, int line){
  if(error != hipSuccess)
  {
    // print the CUDA error message and exit
    fprintf(stderr,"CUDA error: %s in line number %d\n", hipGetErrorString(error),line);
    exit(-1);
  }
}

extern "C" int find_GPUs() {
	
	int deviceCount, device, realGPUs;
	struct hipDeviceProp_t prop;
	hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
	if (cudaResultCode != hipSuccess)
		deviceCount = 0;
		
	// look for the number of real GPUs not counting CUDA emulation devices
	for (device = 0; device < deviceCount; ++device) {
		errorHandler(hipGetDeviceProperties(&prop, device),__LINE__);
		if (prop.major != 9999) { // 9999 is an emulation device
			realGPUs++;
			fprintf(stderr,"Device %d:\n",device);
			fprintf(stderr,"%s @%dMHz\n",prop.name,prop.clockRate/1000);
			fprintf(stderr,"Total memory: %dMBytes\n",prop.totalGlobalMem/(1000*1000));
			fprintf(stderr,"\n");
		}
	}
	
	return realGPUs;
}

// the CUDA compiler generates C++ object files, thus the main procedure has to be an extern "C" for usage in wordom
extern "C" int gClusterRmsd (struct inp_Cluster *inp_cluster,float *distance) {
	
	int ii;
	float cutoff = inp_cluster->threshold;
	int nato = inp_cluster->nato;
	int totframe = inp_cluster->totframe;
	float *gclust_coords = inp_cluster->gclust_coords;
	int *frameapp = inp_cluster->frameapp;
	int super = inp_cluster->super;
	int step = inp_cluster->step;
	int device = inp_cluster->device;
	int frames = totframe/step+(totframe%step == 0 ? 0 : 1); //the number of frames that have to be analysed 
	
	size_t coords_size = 3*nato*sizeof(float); //memory size for coords of a single frame in one dimension
	size_t memsize= frames * coords_size; //memory size for the array of coords of all frames
	size_t cmemsize= frames * sizeof(int); //memory size for the frameapp array
	size_t dmemsize= frames * sizeof(float); //memory size for the distance array
	size_t totalmemsize = memsize + 2*cmemsize + dmemsize;
	
	float *devPtr_gclust_coords;
	float *devPtr_distance;
	int *devPtr_frameapp1;
	int *devPtr_frameapp2;
			
	int threadsPerBlock;
	int blocks;
	struct hipDeviceProp_t properties;
	errorHandler(hipSetDevice(device),__LINE__);
	errorHandler(hipGetDeviceProperties(&properties, device),__LINE__);
		
	fprintf(stderr,"Starting GPU calculation\n");
	
	if(properties.kernelExecTimeoutEnabled)
		fprintf(stderr,"WARNING! The GPU you are using was set up with a run time limit for kernels. Therefore it is likely that the calculation of large proteins fails!\n");
		
	// machines with no GPUs can still report one emulation device 	
	/*for (device = 0; device < deviceCount; ++device) {
		hipGetDeviceProperties(&properties, device);
		if (properties.major != 9999) // 9999 means emulation only
			if (device==0){
				fprintf(stderr,"multiProcessorCount %d\n",properties.multiProcessorCount);
				fprintf(stderr,"maxThreadsPerMultiProcessor %d\n",properties.maxThreadsPerMultiProcessor);
			}
	}*/
	
	//fprintf(stderr,"threads per block: %d\n",threadsPerBlock);
	
	size_t freemem;
	size_t total;
		
	//cuda API functions always return some type of error, but if no error occured, this error is just a hipSuccess
	//errorHandler terminates program in case there was no hipSuccess reported
	errorHandler(hipMemGetInfo(&freemem, &total),__LINE__);
	if(freemem < totalmemsize) {
		fprintf(stderr,"Available graphics memory: %8.3f MBytes\nRequired memory for calculation: %8.3f MBytes\n",(float)freemem/1000000,(float)totalmemsize/1000000);
		fprintf(stderr,"Terminating calculation. Maybe use the --STEP option?\n");
		exit(-1);
	}

	//allocate gpu memory
	errorHandler(hipMalloc((void**)&devPtr_gclust_coords, memsize),__LINE__);
	errorHandler(hipMalloc((void**)&devPtr_distance, dmemsize),__LINE__);
	errorHandler(hipMalloc((void**)&devPtr_frameapp1, cmemsize),__LINE__);
	errorHandler(hipMalloc((void**)&devPtr_frameapp2, cmemsize),__LINE__);
	
	//copy coords to gpu
	errorHandler(hipMemcpy(devPtr_gclust_coords, gclust_coords + 3*nato, frames*coords_size, hipMemcpyHostToDevice),__LINE__);

	//distances are set to the cutoff for the start
	for(ii=0; ii<=frames;ii++) {
			distance[ii] = cutoff;
		}
	errorHandler(hipMemcpy(devPtr_distance, distance+1, dmemsize, hipMemcpyHostToDevice),__LINE__);

	//Set all indices to -1
	errorHandler(hipMemset((void*)devPtr_frameapp1,-1,cmemsize),__LINE__);

	threadsPerBlock = 256; //tuned for 100% occupancy with CUDA occupancy calculator
	blocks = frames/threadsPerBlock +1; //in total we want 1 thread for each frame

	if(super) {

		//shit the center of mass to the origin first for all frames
		shiftToCenter<<<blocks, threadsPerBlock>>>(devPtr_gclust_coords, nato, frames);
		errorHandler( hipPeekAtLastError(),__LINE__);
	
		//change block size for the main kernel function for 100% occupancy again
		if(properties.major == 2)
			threadsPerBlock = 448;
		else
			threadsPerBlock = 256;
			
		blocks = frames/threadsPerBlock +1; //in total we want 1 thread for each frame
				
		for(ii=0;ii< frames;ii++){
						
			//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
			//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
				
			if((ii+1)%2) rmsd_super_kernel<<<blocks, threadsPerBlock>>>(nato, frames, devPtr_gclust_coords, ii, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, inp_cluster->maxspeed);
			else rmsd_super_kernel<<<blocks, threadsPerBlock>>>(nato, frames, devPtr_gclust_coords, ii, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, inp_cluster->maxspeed);
						
			errorHandler( hipPeekAtLastError(),__LINE__);
			fprintf(stderr,"Stage %% %f\r",(float)ii/frames*100.0);//just a progress bar
						
		}
		
		
	} else {
		
		for(ii=0;ii< frames;ii++){
						
			//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
			//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
				
			if((ii+1)%2) rmsd_kernel<<<blocks, threadsPerBlock>>>(nato, frames, devPtr_gclust_coords, ii, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, inp_cluster->maxspeed);
			else rmsd_kernel<<<blocks, threadsPerBlock>>>(nato, frames, devPtr_gclust_coords, ii, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, inp_cluster->maxspeed);
						
			errorHandler( hipPeekAtLastError(),__LINE__);
			fprintf(stderr,"Stage %% %f\r",(float)ii/frames*100.0);//just a progress bar
						
		}
	}	
		printf("\n");
			
		//DEBUG fprintf(stderr,"Copying results to Host ..\n");
		
		//make sure to copy the correct frameapp array back
		if((ii+1)%2) errorHandler( hipMemcpy(frameapp+1, devPtr_frameapp1, cmemsize, hipMemcpyDeviceToHost),__LINE__);
		else errorHandler( hipMemcpy(frameapp+1, devPtr_frameapp2, cmemsize, hipMemcpyDeviceToHost),__LINE__);
			
		errorHandler( hipMemcpy(distance+1, devPtr_distance, dmemsize, hipMemcpyDeviceToHost),__LINE__);
			
		//free GPU memory
		errorHandler( hipFree(devPtr_gclust_coords),__LINE__);
		errorHandler( hipFree(devPtr_frameapp1),__LINE__);
		errorHandler( hipFree(devPtr_frameapp2),__LINE__);
		errorHandler( hipFree(devPtr_distance),__LINE__);		
		return 0;		
}
	

// the CUDA compiler generates C++ object files, thus the main procedure has to be an extern "C" for usage in wordom
extern "C" int gClusterDrms (struct inp_Cluster *inp_cluster,float *distance)
{
    int ii;
    float cutoff = inp_cluster->threshold;
   	int msize = inp_cluster->msize;
	int totframe = inp_cluster->totframe;
	float *gclust_dmtx = inp_cluster->gclust_dmtx;
    float nointrasegm_corr_fact = 1.0;
    int *frameapp = inp_cluster->frameapp;
    int step = inp_cluster->step;
    int device = inp_cluster->device;
    int frames = totframe/step+(totframe%step == 0 ? 0 : 1); //the number of frames that have to be analysed 
     
    //change correction factor if correction should be applied
	if( inp_cluster->nointrasegm != 0)
		nointrasegm_corr_fact = inp_cluster->nointrasegm_corr_fact;

	size_t dmtx_size = msize*sizeof(float); //memory size for a single distance matrix
	size_t memsize= frames * dmtx_size; //memory size for the array of distance matrices
	size_t cmemsize= frames * sizeof(int); //memory size for the frameapp array
	size_t dmemsize= frames * sizeof(float); //memory size for the distance array
	size_t totalmemsize = memsize + cmemsize + dmemsize;
	
	float *devPtr_gclust_dmtx;
	float *devPtr_distance;
	int *devPtr_frameapp1;
	int *devPtr_frameapp2;
				
	// machines with no GPUs can still report one emulation device 	
	/*for (device = 0; device < deviceCount; ++device) {
		hipGetDeviceProperties(&properties, device);
		if (properties.major != 9999) // 9999 means emulation only
			if (device==0){
				fprintf(stderr,"multiProcessorCount %d\n",properties.multiProcessorCount);
				fprintf(stderr,"maxThreadsPerMultiProcessor %d\n",properties.maxThreadsPerMultiProcessor);
				
				if(properties.major == 2)
					threadsPerBlock = 192;
				else
					threadsPerBlock = 256;
			}
	}*/
	
	struct hipDeviceProp_t properties;
	errorHandler(hipSetDevice(device),__LINE__);
	errorHandler(hipGetDeviceProperties(&properties, device),__LINE__);
		
	fprintf(stderr,"Starting GPU calculation\n");
	
	if(properties.kernelExecTimeoutEnabled)
		fprintf(stderr,"WARNING! The GPU you are using was set up with a run time limit for kernels. Therefore it is likely that the calculation of large proteins fails!\n");
		
	int threadsPerBlock;
	if(properties.major == 2)
		threadsPerBlock = 192;
	else
		threadsPerBlock = 256;
	
	size_t freemem;
	size_t total;
		
	//cuda API functions always return some type of error, but if no error occured, this error is just a hipSuccess
	//errorHandler terminates program in case there was no hipSuccess reported
	errorHandler(hipMemGetInfo(&freemem, &total),__LINE__);
	
	//check if there is enough gpu memory for the job and split up the calculation if not
	if(freemem < totalmemsize) {
		//DEBUG fprintf(stderr,"Available memory on device: %u\n Total memory necessary on device for calculation: %u\n .. splitting up calculation\n",freemem,totalmemsize);
		
		int framesRemaining = frames; //the number of frames that still have to be analysed
		int framesFinished = 0;
		int nclusters = 0; //the number of clusters already found
		int newClusters = 0;
		int *cluster = (int*)calloc(frames,sizeof(int)); //clustercenters of existing clusters
		int *devPtr_cluster; //stores the clustercenter of new found clusters
		int *devPtr_newClusters; //stores the number of new found clusters
		int nframes;	

		while(framesRemaining > 0) {
				
			size_t clust_dmtx_mem = nclusters * dmtx_size; //additional memory for the clusters' distance matrices
			errorHandler(hipMemGetInfo(&freemem, &total),__LINE__);
			
			//number of frames that fit into memory; 2MB of the total memory reported to freemem have to remain free, allocations fail otherwise (value found by trial and error)
			nframes = (freemem -10000000 - clust_dmtx_mem - sizeof(int))/(dmtx_size+3*sizeof(int)+sizeof(float));
			//DEBUG fprintf(stderr,"Free memory: %u, Frames remaining: %d, Frames fitting into memory: %d, Number of clusters: %d\n",freemem,framesRemaining,nframes,nclusters);
			
			//nframes is either the number of frames that fit into gpu memory, or the number of remaining frames
			if(nframes >= framesRemaining) {
				nframes = framesRemaining;
			} else {
				
				//if the number of clusters gets too high the calculation has to be stopped
				if(nclusters > nframes) {
					fprintf(stderr,"Number of clusters has exceeded number of frames that fit on GPU memory, calculation is getting too slow!\n Quitting calculation... Please choose a greater cutoff!\n");
					exit(-1);
				}
			}		
			
			//recalculate the memory sizes
			size_t memsize= nframes * dmtx_size;
			size_t cmemsize= nframes * sizeof(int);
			size_t dmemsize= nframes * sizeof(float);	
				
			//allocating memory on the GPU
			errorHandler(hipMalloc((void**)&devPtr_gclust_dmtx,memsize+clust_dmtx_mem),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_distance,dmemsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_frameapp1,cmemsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_frameapp2,cmemsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_cluster,cmemsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_newClusters,sizeof(int)),__LINE__);
					
			//if there were already clusters found, copy their distance matrices
			if(clust_dmtx_mem>0){
				//because of the overhead of a single copy instruction we prefer to copy one large data packet over lots of small ones, we use a temporary array for this
				float *clusters_dmtx;
				clusters_dmtx=(float *)malloc(clust_dmtx_mem);
				
				for(ii = 0; ii < nclusters; ii++)
					memcpy(clusters_dmtx + ii*msize,gclust_dmtx + cluster[ii]*msize,dmtx_size);
				
				errorHandler(hipMemcpy(devPtr_gclust_dmtx,clusters_dmtx,clust_dmtx_mem,hipMemcpyHostToDevice),__LINE__);
				free(clusters_dmtx);
			}		
			
			//copy distance matrices of the frames to gpu, they are copied right after the distance matrices of the clusters
			errorHandler(hipMemcpy(devPtr_gclust_dmtx + nclusters*msize, gclust_dmtx + (framesFinished + 1)*msize, memsize, hipMemcpyHostToDevice),__LINE__);	
				
			//Sets all indices to -1
			errorHandler(hipMemset((void*)devPtr_frameapp1,-1,cmemsize),__LINE__);
			
			//in order to find the closest cluster we set the distances to the cutoff for the start
			if(!inp_cluster->maxspeed){
				for(ii=framesFinished; ii<=nframes+framesFinished;ii++) {
					distance[ii] = cutoff;
				}
				errorHandler(hipMemcpy(devPtr_distance, distance+framesFinished+1, dmemsize, hipMemcpyHostToDevice),__LINE__);
			}	
				
			//set number of new clusters to 0
			errorHandler(hipMemset((void*)devPtr_newClusters,0,sizeof(int)),__LINE__);
					
			int blocks = nframes/threadsPerBlock +1; //in total we want 1 thread for each frame
						
			if(inp_cluster->maxspeed) {			
				//compare the frames of the chunk to the previously found clusters first
				for(ii = 0; ii < nclusters; ii++) {		
							
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					if((ii+1)%2) gDrmsClustersMax<<<blocks, threadsPerBlock>>>(msize, nframes, devPtr_gclust_dmtx, ii, cutoff, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, nointrasegm_corr_fact, nclusters, cluster[ii]);
					else gDrmsClustersMax<<<blocks, threadsPerBlock>>>(msize, nframes, devPtr_gclust_dmtx, ii, cutoff, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, nointrasegm_corr_fact, nclusters, cluster[ii]);
					
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Comparing to previous clusters %% %f\r",(double)ii/nclusters*100.0);//just a progress bar
									
				}
				
				fprintf(stderr,"\n");
				
				//then check the remaining frames against each other
				for(ii=0; ii < nframes; ii++){	
						
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					if((nclusters + ii + 1)%2) gDrmsFramesMax<<<blocks, threadsPerBlock>>>(msize, nframes, devPtr_gclust_dmtx,ii, cutoff, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, nointrasegm_corr_fact, framesFinished, nclusters, devPtr_newClusters, devPtr_cluster);
					else gDrmsFramesMax<<<blocks, threadsPerBlock>>>(msize, nframes, devPtr_gclust_dmtx, ii, cutoff, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, nointrasegm_corr_fact, framesFinished, nclusters, devPtr_newClusters, devPtr_cluster);
					
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Calculating Stage %% %f\r",(double)(framesFinished + ii)/frames*100.0);//just a progress bar
					
				}
				
			} else {	
				//compare the frames of the chunk to the previously found clusters first						
				for(ii = 0; ii < nclusters; ii++) {		
					
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					if((ii+1)%2) gDrmsClustersClosest<<<blocks, threadsPerBlock>>>(msize, nframes, devPtr_gclust_dmtx, ii, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, nointrasegm_corr_fact, nclusters, cluster[ii]);
					else gDrmsClustersClosest<<<blocks, threadsPerBlock>>>(msize, nframes, devPtr_gclust_dmtx, ii, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, nointrasegm_corr_fact, nclusters, cluster[ii]);
					
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Comparing to previous clusters %% %f\r",(double)ii/nclusters*100.0);//just a progress bar
					
				}
				
				fprintf(stderr,"\n");
				
				//then check the remaining frames against each other	
				for(ii=0; ii < nframes; ii++){
					
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					if((nclusters + ii + 1)%2) gDrmsFramesClosest<<<blocks, threadsPerBlock>>>(msize, nframes, devPtr_gclust_dmtx, ii, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, nointrasegm_corr_fact, framesFinished, nclusters, devPtr_newClusters, devPtr_cluster);
					else gDrmsFramesClosest<<<blocks, threadsPerBlock>>>(msize, nframes, devPtr_gclust_dmtx, ii, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, nointrasegm_corr_fact, framesFinished, nclusters, devPtr_newClusters, devPtr_cluster);
					
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Calculating Stage %% %f\r",(double)(framesFinished + ii)/frames*100.0);//just a progress bar
					
				}
			}
			
			printf("\n");	
			//DEBUG fprintf(stderr,"Copying to Host ..\n");
			
			//copy back to host, by adding framesFinished/nclusters to the pointers we make sure not to overwrite the results from previous runs
			if((nclusters + ii + 1)%2) errorHandler( hipMemcpy(frameapp+framesFinished + 1, devPtr_frameapp1, cmemsize, hipMemcpyDeviceToHost),__LINE__);
			else errorHandler( hipMemcpy(frameapp+framesFinished + 1, devPtr_frameapp2, cmemsize, hipMemcpyDeviceToHost),__LINE__);
			errorHandler( hipMemcpy(distance+framesFinished + 1, devPtr_distance, dmemsize, hipMemcpyDeviceToHost),__LINE__);
			errorHandler( hipMemcpy(&newClusters, devPtr_newClusters, sizeof(int), hipMemcpyDeviceToHost),__LINE__);
			errorHandler( hipMemcpy(cluster+nclusters, devPtr_cluster,newClusters*sizeof(int),hipMemcpyDeviceToHost),__LINE__);
									
			//update number of clusters and processed frames
			nclusters += newClusters;
			framesFinished += nframes;
			framesRemaining -= nframes;	
			
			//free all the GPU memory
			errorHandler(hipFree(devPtr_gclust_dmtx),__LINE__);
			errorHandler(hipFree(devPtr_distance),__LINE__);
			errorHandler(hipFree(devPtr_frameapp1),__LINE__);
			errorHandler(hipFree(devPtr_frameapp2),__LINE__);
			errorHandler(hipFree(devPtr_cluster),__LINE__);
			errorHandler(hipFree(devPtr_newClusters),__LINE__);			
		}	
		
			
		return 0;		
		
		} else {
			
			//allocate gpu memory
			errorHandler(hipMalloc((void**)&devPtr_gclust_dmtx, memsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_distance, dmemsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_frameapp1, cmemsize),__LINE__); 
			errorHandler(hipMalloc((void**)&devPtr_frameapp2, cmemsize),__LINE__); 
					
			//copy distance matrices to gpu
			errorHandler(hipMemcpy(devPtr_gclust_dmtx, gclust_dmtx + msize, memsize, hipMemcpyHostToDevice),__LINE__);
		
			//Set all indices to -1
			errorHandler(hipMemset((void*)devPtr_frameapp1,-1,cmemsize),__LINE__);
			
			//in order to find the closest cluster we set the distances to the cutoff for the start
			if(!inp_cluster->maxspeed){
				for(ii=0; ii<=frames;ii++) {
					distance[ii] = cutoff;
				}
				errorHandler(hipMemcpy(devPtr_distance, distance+1, dmemsize, hipMemcpyHostToDevice),__LINE__);
			}

			int blocks = frames/threadsPerBlock +1; //in total we want 1 thread for each frame
				
			if(inp_cluster->maxspeed) {
				for(ii=0;ii< frames;ii++){
					
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					if((ii+1)%2) gDrmsMax<<<blocks, threadsPerBlock>>>(msize, frames, devPtr_gclust_dmtx,ii, cutoff, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, nointrasegm_corr_fact);
					else gDrmsMax<<<blocks, threadsPerBlock>>>(msize, frames, devPtr_gclust_dmtx,ii, cutoff, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, nointrasegm_corr_fact);
					
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Stage %% %f\r",(double)ii/frames*100.0);//just a progress bar
					
				}
				
			} else {
				for(ii=0;ii< frames;ii++){
					
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					if((ii+1)%2) gDrmsClosest<<<blocks, threadsPerBlock>>>(msize, frames, devPtr_gclust_dmtx, ii, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, nointrasegm_corr_fact);
					else gDrmsClosest<<<blocks, threadsPerBlock>>>(msize, frames, devPtr_gclust_dmtx, ii, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, nointrasegm_corr_fact);
					
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Stage %% %f\r",(double)ii/frames*100.0);//just a progress bar
					
				}	
			}	
			printf("\n");
			
			//DEBUG fprintf(stderr,"Copying results to Host ..\n");
		
			if((ii+1)%2) errorHandler( hipMemcpy(frameapp+1, devPtr_frameapp1, cmemsize, hipMemcpyDeviceToHost),__LINE__);
			else errorHandler( hipMemcpy(frameapp+1, devPtr_frameapp2, cmemsize, hipMemcpyDeviceToHost),__LINE__);
			
			errorHandler( hipMemcpy(distance+1, devPtr_distance, dmemsize, hipMemcpyDeviceToHost),__LINE__);
			
			//free GPU memory
			errorHandler( hipFree(devPtr_gclust_dmtx),__LINE__);
			errorHandler( hipFree(devPtr_frameapp1),__LINE__);
			errorHandler( hipFree(devPtr_frameapp2),__LINE__);
			errorHandler( hipFree(devPtr_distance),__LINE__);		
			return 0;
	   }
	}
