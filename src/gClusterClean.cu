#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h> 
#include <stdlib.h>
	//including wordom.h generates compiler warnings
	//we do not have to worry about this, since this only happens because nvcc generates c++ object files and the c++ compiler is not completely satisfied with some stuff in wordom.h
#include "wordom.h"
#include "fileio.h"
#include "tools.h"
#include "analysis.h"
#include "cluster.h"

__global__ void gRmsdMax(const int msize, const int nframes, const float* gclust_coords, const int cluster, const int* frameapp_read, int* frameapp_write, float* distance) {
	
	
	// nato = number of atoms, nframes = number of frames used for clustering;
	// gclust_coords = array of all coordinates of the frames; cluster = center of a possible cluster; frameapp stores the cluster center 
	// of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays;
	// distance stores the calculated rmsd because it will be needed in the post processing
	

	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	//with the maxspeed flag frames are assigned to the first cluster -> do not check frame again if it was already assigned
	if(frameapp_read[index] != -1) {
		frameapp_write[index] = frameapp_read[index];
		return;
	}
	
	//if cluster has already been assigned to another cluster then it cannot be a new cluster
	if(frameapp_read[cluster] == -1) {
		
		//spare the distance calculation if index and cluster are the same, by wordoms conventions the distance is set to -1 instead of 0 in this case
		if(cluster == index) {
			frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = -1.0;
			return;
		}
		
		float rmsd=0. , di;
		int ii;
		
	    
	    // compute rmsd and return value
			  
		for ( ii=0; ii<msize; ii++ ) {
				di=gclust_coords[index*msize+ii]-gclust_coords[cluster*msize+ii];
				rmsd+=di*di;
				/*di=gclust_coords[nframes*nato+index*nato+ii]-gclust_coords[nframes*nato+cluster*nato+ii];
				rmsd+=di*di;
				di=gclust_coords[2*nframes*nato+index*nato+ii]-gclust_coords[2*nframes*nato+cluster*nato+ii];
				rmsd+=di*di;*/
			}
			  
		rmsd /= nato;
		rmsd = sqrt ( rmsd );
	
		if (rmsd<distance[index]){		
			frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = rmsd;
			return;
		}
		
	}
	frameapp_write[index] = frameapp_read[index];
			  
			
}

__global__ void gRmsdClosest(const int msize, const int nframes, const float* gclust_coords, const int cluster, const float* frameapp_read, float* frameapp_write, float* distance) {
	
	// nato = number of atoms; nframes = number of frames used for clustering;
	// gclust_coords = array of all coordinates of the frames; cluster = center of a possible cluster; frameapp stores the cluster center 
	// of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays;
	// distance stores the calculated drms for comparison
	

	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	//frames can only be assigned to clusters with a lower frame number
	if(cluster > index) {
		frameapp_write[index] = frameapp_read[index];
		return;
	}
	
	//if cluster has already been assigned to another cluster then it cannot be a new cluster
	if(frameapp_read[cluster] == -1) {
	
		//spare the distance calculation if index and cluster are the same, by wordoms conventions the distance is set to -1 instead of 0 in this case
		if(cluster == index) {
			frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = -1.0;
			return;
		}		
		float rmsd=0. , di;
		int jj;
		
		//calculate the drms of cluster and index
		for (jj=0;jj<msize;jj++){
			di=(gclust_coords[index*msize+jj]-gclust_coords[cluster*msize+jj]);
			rmsd+=di*di; 
		}			
		rmsd /= nato;
		rmsd = sqrt ( rmsd );
	
		//at the beginning distance is set to the cutoff, by always comparing the drms to the current value of distance instead of only the cutoff we can reassign the frame if we find a closer cluster
		if (rmsd < distance[index]){		
			frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = rmsd;
			return;
		}
	}
	
	frameapp_write[index] = frameapp_read[index];
}


__global__ void gRmsdClustersMax(const int msize, const int nframes, const float* gclust_coords, const int cluster, const int* frameapp_read, int* frameapp_write, float* distance, const int nclusters, const int clustercenter) {
	

	// nato = number of atoms; nframes = number of frames used for clustering;
	// gclust_coords = array of all coordinates of the frames; cluster = number of the cluster; frameapp stores the cluster center 
	// of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays; 
	// distance stores the calculated drms because it will be needed in the post processing;
	// nclusters passes the number of already found clusters; clustercenter passes the center of the current cluster

	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x; 	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	//with maxspeed frames are assigned to the first cluster -> do not check frame again if it was already assigned
	if(frameapp_read[index] != -1) {
		frameapp_write[index] = frameapp_read[index];
		return;
	}		

	float rmsd=0. , di;
	int ii;
	
	    // compute rmsd and return value			  

	for ( ii=0; ii<msize; ii++ ) {
			di=gclust_coords[(index+nclusters)*msize+ii]-gclust_coords[cluster*msize+ii];
			rmsd+=di*di;
			/*di=gclust_coords[nframes*nato+index*nato+ii]-gclust_coords[nframes*nato+cluster*nato+ii];
			rmsd+=di*di;
			di=gclust_coords[2*nframes*nato+index*nato+ii]-gclust_coords[2*nframes*nato+cluster*nato+ii];
			rmsd+=di*di;*/
		}
			  
		rmsd /= nato;
		rmsd = sqrt ( rmsd );

		if (rmsd<distance[index]){		
			frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = rmsd;
			return;
		}
	
	frameapp_write[index] = frameapp_read[index];
}

__global__ void gRmsdFramesMax(const int msize, const int nframes, const float* gclust_coords, const int cluster, const int* frameapp_read, int* frameapp_write, float* distance, const int framesFinished, const int nclusters, int* newClusters, int* cluster) {
	
	
	// nato = number of atoms; nframes = number of frames used for clustering;
	// gclust_coords = array of all coordinates of the frames; cluster = center of a possible cluster; frameapp stores the cluster center 
	// of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays;
	// distance stores the calculated drms because it will be needed in the post processing;
	// framesFinished passes the number of processed frames in a previous chunk; nclusters passes the number of already found clusters;
	// newClusters stores the number of new found clusters in this chunk; clusterCenters stores the centers of these clusters
	

	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	//with maxspeed frames are assigned to the first cluster -> do not check frame again if it was already assigned
	if(frameapp_read[index] != -1) {
		frameapp_write[index] = frameapp_read[index];
		return;
	}
	
	//if cluster has already been assigned to another cluster then it cannot be a new cluster
	if(frameapp_read[cluster] == -1) {
		
		//spare the distance calculation if index and cluster are the same, by wordoms conventions the distance is set to -1 instead of 0 in this case
		if(cluster == index) {
			frameapp_write[index] = framesFinished + cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = -1.0;
			clusterCenters[*newClusters] = framesFinished + cluster + 1;
			(*newClusters)++;
			return;
		}
		
		float rmsd=0. , di;
		int jj;
		
		//calculate the drms of cluster and index
		for (jj=0;jj<msize;jj++){
			di=gclust_coords[(index+nclusters)*msize+jj]-gclust_coords[(cluster+nclusters)*msize+jj]);
			rmsd+=di*di; 
		}
					
		rmsd /= nato;
		rmsd = sqrt ( rmsd );
	
		if (rmsd<distance[index]){		
			frameapp_write[index] = framesFinished + cluster + 1;
			distance[index] = rmsd;
			return;
		}

	}
	frameapp_write[index] = frameapp_read[index];
}


__global__ void gRmsdClustersClosest(const int nato, const int nframes, const float* gclust_coords, const int cluster, const int* frameapp_read, int* frameapp_write, float* distance, const int nclusters, const int clustercenter) {
	
	// nato = number of atoms; nframes = number of frames used for clustering;
	// gclust_coords = array of all coordinates of the frames; cluster = number of the cluster; frameapp stores the cluster center 
	// of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays; 
	// distance stores the calculated for comparison;
	// nclusters passes the number of already found clusters; clustercenter passes the center of the current cluster
	

	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	float rmsd=0. , di;
	int jj;
	
	//calculate the rmsd of cluster and index
	for (jj=0;jj<3*nato;jj++){
		di=(gclust_coords[(index+nclusters)*3*nato+jj]-gclust_coords[cluster*3*nato+jj]);
		drms+=di*di; 
	}
		
	rmsd /= nato;
	rmsd = sqrt ( rmsd );

	//at the beginning distance is set to cutoff, by always comparing the drms to the current value of distance instead of the cutoff we can reassign the frame if we find a closer cluster
	if (rmsd < distance[index]){		
		frameapp_write[index] = clustercenter;
		distance[index] = rmsd;
		return;
	}
	frameapp_write[index] = frameapp_read[index];
	
}
__global__ void gRmsdFramesClosest(const int nato, const int nframes, const float* gclust_coords, const int cluster, const int* frameapp_read, int* frameapp_write, float* distance, const int framesFinished, const int nclusters, int* newClusters, int* clusterCenters) {
	
	// nato = number of atoms; nframes = number of frames used for clustering;
	// gclust_coords = array of all coordinates of the frames; cluster = center of a possible cluster; frameapp stores the cluster center 
	// of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays;
	// distance stores the calculated drms for comparison;
	// framesFinished passes the number of processed frames in a previous chunk; nclusters passes the number of already found clusters;
	// newClusters stores the number of new found clusters in this chunk; clusterCenters stores the centers of these clusters	
	
	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	//frames can only be assigned to clusters with a lower frame number
	if(cluster > index) {
		frameapp_write[index] = frameapp_read[index];
		return;
	}
	
	//if cluster has already been assigned to another cluster then it cannot be a new cluster
	if(frameapp_read[cluster] == -1) {
	
		//spare the distance calculation if index and cluster are the same, by wordoms conventions the distance is set to -1 instead of 0 in this case
		if(cluster == index) {
			frameapp_write[index] = framesFinished + cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = -1.0;
			clusterCenters[*newClusters] = framesFinished + cluster + 1;
			(*newClusters)++;
			return;
		}		
		float rmsd=0. , di;
		int jj;
		
		//calculate the drms of cluster and index
		for (jj=0;jj<3*nato;jj++){
			di=(gclust_coords[(index+nclusters)*3*nato+jj]-gclust_dmtx[(cluster+nclusters)*3*nato+jj]);
			rmsd+=di*di; 
		}			
		rmsd /= nato;
		rmsd = sqrt ( rmsd );
	
		//at the beginning distance is set to cutoff, by always comparing the drms to the current value of distance instead of the cutoff we can reassign the frame if we find a closer cluster
		if (rmsd < distance[index]){	
			frameapp_write[index] = framesFinished + cluster + 1;
			distance[index] = rmsd;
			return;
		}	
	}
	frameapp_write[index] = frameapp_read[index];
	
}

// a kernel for the maxspeed flag
__global__ void gDrms(const int max, const int msize, const int nframes, const float* gclust_dmtx, const int cluster, const int* frameapp_read, int* frameapp_write, float* distance, const float nointrasegm_corr_fact) {
	
	// max determines to run either the maxspeed (=1) or closest cluster search (=0)
	// msize = size of each distance matrix, nframes = number of frames used for clustering;
	// gclust_dmtx = array of ALL distance matrices; cluster = center of a possible cluster; frameapp stores the cluster center 
	// of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays;
	// distance stores the calculated distance for comparison and later usage
	

	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	if(max) {
		//with the maxspeed flag frames are assigned to the first cluster -> do not check frame again if it was already assigned
		if(frameapp_read[index] != -1) {
			frameapp_write[index] = frameapp_read[index];
			return;
		} 
	} else {
	
		//frames can only be assigned to clusters with a lower frame number
		if(cluster > index) {
			frameapp_write[index] = frameapp_read[index];
			return;
		}
	}
	
	//if cluster has already been assigned to another cluster then it cannot be a new cluster
	if(frameapp_read[cluster] == -1) {
		
		//spare the distance calculation if index and cluster are the same, by wordoms conventions the distance is set to -1 instead of 0 in this case
		if(cluster == index) {
			frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = -1.0;
			return;
		}
		
		float drms=0. , di;
		int jj;
		
		//calculate the drms of cluster and index
		for (jj=0;jj<msize;jj++){
			di=(gclust_dmtx[index*msize+jj]-gclust_dmtx[cluster*msize+jj]);
			drms+=di*di; 
		}		
		drms = sqrtf(drms);
	    drms *= nointrasegm_corr_fact/sqrtf((float)msize); //Renormalize the distance properly
	
		//at the beginning distance is set to the cutoff, by always comparing the drms to the current value of distance instead of only the cutoff we can reassign the frame if we find a closer cluster
		if (drms<distance[index]){		
			frameapp_write[index] = cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = drms;
			return;
		}
		
	}
	frameapp_write[index] = frameapp_read[index];

}

// a kernel for the maxspeed flag for calculation with limited memory; this is for comparing the frames of the chunk to the previously found clusters
__global__ void gDrmsClustersMax(const int msize, const int nframes, const float* gclust_dmtx, const int cluster, const int* frameapp_read, int* frameapp_write, float* distance, const float nointrasegm_corr_fact, const int nclusters, const int clustercenter) {
	
	// msize = size of each distance matrix; nframes = number of frames used for clustering;
	// gclust_dmtx = array of ALL distance matrices; cluster = number of the cluster; frameapp stores the cluster center 
	// of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays; 
	// distance stores the calculated drms because it will be needed in the post processing;
	// nclusters passes the number of already found clusters; clustercenter passes the center of the current cluster

	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x; 	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	//with maxspeed frames are assigned to the first cluster -> do not check frame again if it was already assigned
	if(frameapp_read[index] != -1) {
		frameapp_write[index] = frameapp_read[index];
		return;
	}		
	float drms=0. , di;
	int jj;
	
	//calculate the drms of cluster and index
	for (jj=0;jj<msize;jj++){
		di=(gclust_dmtx[(index+nclusters)*msize+jj]-gclust_dmtx[cluster*msize+jj]);
		drms+=di*di; 
	}			
	drms = sqrtf(drms);
    drms *= nointrasegm_corr_fact/sqrtf(msize); //Renormalize the distance properly

	if (drms<distance[index]){		
		frameapp_write[index] = clustercenter;
		distance[index] = drms;
		return;
	}
	
	frameapp_write[index] = frameapp_read[index];
}

// a kernel for the lfull flag for calculation with limited memory; this is for comparing the frames of the chunk to the previously found clusters
__global__ void gDrmsClustersClosest(const int msize, const int nframes, const float* gclust_dmtx, const int cluster, const int* frameapp_read, int* frameapp_write, float* distance, const float nointrasegm_corr_fact, const int nclusters, const int clustercenter) {
	
	// msize = size of each distance matrix; nframes = number of frames used for clustering;
	// gclust_dmtx = array of ALL distance matrices; cluster = number of the cluster; frameapp stores the cluster center 
	// of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays; 
	// distance stores the calculated for comparison;
	// nclusters passes the number of already found clusters; clustercenter passes the center of the current cluster

	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}		
	float drms=0. , di;
	int jj;
	
	//calculate the drms of cluster and index
	for (jj=0;jj<msize;jj++){
		di=(gclust_dmtx[(index+nclusters)*msize+jj]-gclust_dmtx[cluster*msize+jj]);
		drms+=di*di; 
	}			
	drms = sqrtf(drms);
    drms *= nointrasegm_corr_fact/sqrtf(msize); //Renormalize the distance properly

	//at the beginning distance is set to cutoff, by always comparing the drms to the current value of distance instead of the cutoff we can reassign the frame if we find a closer cluster
	if (drms < distance[index]){		
		frameapp_write[index] = clustercenter;
		distance[index] = drms;
		return;
	}
	frameapp_write[index] = frameapp_read[index];
}

// a kernel for the maxspeed flag for calculation with limited memory; this is for comparing the frames of the chunk among themselves
__global__ void gDrmsFramesMax(const int msize, const int nframes, const float* gclust_dmtx, const int cluster, const int* frameapp_read, int* frameapp_write, float* distance, const float nointrasegm_corr_fact, const int framesFinished, const int nclusters, int* newClusters, int* clusterCenters ) {
	
	// msize = size of each distance matrix; nframes = number of frames used for clustering;
	// gclust_dmtx = array of ALL distance matrices; cluster = center of a possible cluster; frameapp stores the cluster center 
	// of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays;
	// distance stores the calculated drms because it will be needed in the post processing;
	// framesFinished passes the number of processed frames in a previous chunk; nclusters passes the number of already found clusters;
	// newClusters stores the number of new found clusters in this chunk; clusterCenters stores the centers of these clusters
	

	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	//with maxspeed frames are assigned to the first cluster -> do not check frame again if it was already assigned
	if(frameapp_read[index] != -1) {
		frameapp_write[index] = frameapp_read[index];
		return;
	}
	
	//if cluster has already been assigned to another cluster then it cannot be a new cluster
	if(frameapp_read[cluster] == -1) {
		
		//spare the distance calculation if index and cluster are the same, by wordoms conventions the distance is set to -1 instead of 0 in this case
		if(cluster == index) {
			frameapp_write[index] = framesFinished + cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = -1.0;
			clusterCenters[*newClusters] = framesFinished + cluster + 1;
			(*newClusters)++;
			return;
		}		
		float drms=0. , di;
		int jj;
		
		//calculate the drms of cluster and index
		for (jj=0;jj<msize;jj++){
			di=(gclust_dmtx[(index+nclusters)*msize+jj]-gclust_dmtx[(cluster+nclusters)*msize+jj]);
			drms+=di*di; 
		}			
		drms = sqrtf(drms);
	    drms *= nointrasegm_corr_fact/sqrtf((float)msize); //Renormalize the distance properly
	
		if (drms<distance[index]){		
			frameapp_write[index] = framesFinished + cluster + 1;
			distance[index] = drms;
			return;
		}
	}
	frameapp_write[index] = frameapp_read[index];
}

// a kernel for the lfull flag for calculation with limited memory; this is for comparing the frames of the chunk among themselves
__global__ void gDrmsFramesClosest(const int msize, const int nframes, const float* gclust_dmtx, const int cluster, const int* frameapp_read, int* frameapp_write, float* distance, const float nointrasegm_corr_fact, const int framesFinished, const int nclusters, int* newClusters, int* clusterCenters ) {
	
	// msize = size of each distance matrix; nframes = number of frames used for clustering;
	// gclust_dmtx = array of ALL distance matrices; cluster = center of a possible cluster; frameapp stores the cluster center 
	// of each frame. There is one frameapp array for reading only and one for writing only, because otherwise this could go wrong for very long arrays;
	// distance stores the calculated drms for comparison;
	// framesFinished passes the number of processed frames in a previous chunk; nclusters passes the number of already found clusters;
	// newClusters stores the number of new found clusters in this chunk; clusterCenters stores the centers of these clusters	
	
	// threadIdx.x is a built-in variable provided by CUDA at runtime, this gives each of the threads a unique index that corresponds to a frame number 
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	// if there are more threads than frames then stop these
	if (index>=nframes){
		return;
	}
	
	//frames can only be assigned to clusters with a lower frame number
	if(cluster > index) {
		frameapp_write[index] = frameapp_read[index];
		return;
	}
	
	//if cluster has already been assigned to another cluster then it cannot be a new cluster
	if(frameapp_read[cluster] == -1) {
	
		//spare the distance calculation if index and cluster are the same, by wordoms conventions the distance is set to -1 instead of 0 in this case
		if(cluster == index) {
			frameapp_write[index] = framesFinished + cluster + 1; //+1 because in wordom the frames are counted starting with 1
			distance[index] = -1.0;
			clusterCenters[*newClusters] = framesFinished + cluster + 1;
			(*newClusters)++;
			return;
		}		
		float drms=0. , di;
		int jj;
		
		//calculate the drms of cluster and index
		for (jj=0;jj<msize;jj++){
			di=(gclust_dmtx[(index+nclusters)*msize+jj]-gclust_dmtx[(cluster+nclusters)*msize+jj]);
			drms+=di*di; 
		}			
		drms = sqrtf(drms);
	    drms *= nointrasegm_corr_fact/sqrtf((float)msize); //Renormalize the distance properly
	
		//at the beginning distance is set to cutoff, by always comparing the drms to the current value of distance instead of the cutoff we can reassign the frame if we find a closer cluster
		if (drms < distance[index]){	
			frameapp_write[index] = framesFinished + cluster + 1;
			distance[index] = drms;
			return;
		}	
	}
	frameapp_write[index] = frameapp_read[index];
}

//A macro for handling CUDA errors
void errorHandler  (hipError_t error, int line){
  if(error != hipSuccess)
  {
    // print the CUDA error message and exit
    fprintf(stderr,"CUDA error: %s in line number %d\n", hipGetErrorString(error),line);
    exit(-1);
  }
}

// the CUDA compiler generates C++ object files, thus the main procedure has to be an extern "C" for usage in wordom
extern "C" int gClusterRmsd (struct inp_Cluster *inp_cluster,float *distance) {
	
	
	int ii;
    float cutoff = inp_cluster->threshold;
   	int nato = inp_cluster->nato;
	int totframe = inp_cluster->totframe;
	//float **gclust_coords = inp_cluster->gclust_coords;
	float *gclust_coords = inp_cluster->gclust_coords;
    int *frameapp = inp_cluster->frameapp;
    int step = inp_cluster->step;
    int frames = totframe/step+(totframe%step == 0 ? 0 : 1); //the number of frames that have to be analysed 

	size_t coords_size = 3*nato*sizeof(float); //memory size for coords of a single frame in one dimension
	size_t memsize= frames * coords_size; //memory size for the array of coords of all frames
	size_t cmemsize= frames * sizeof(int); //memory size for the frameapp array
	size_t dmemsize= frames * sizeof(float); //memory size for the distance array
	size_t totalmemsize = memsize + 2*cmemsize + dmemsize;
	
	float *devPtr_gclust_coords;
	float *devPtr_distance;
	int *devPtr_frameapp1;
	int *devPtr_frameapp2;
			
	int deviceCount; // number of devices, i.e. gpus
	int device;
	int threadsPerBlock;
	struct hipDeviceProp_t properties;		
	hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
	if (cudaResultCode != hipSuccess)
		deviceCount = 0;
		
	fprintf(stderr,"Starting GPU calculation, devicecount : %d\n", deviceCount);
		
	// machines with no GPUs can still report one emulation device 	
	for (device = 0; device < deviceCount; ++device) {
		hipGetDeviceProperties(&properties, device);
		if (properties.major != 9999) // 9999 means emulation only
			if (device==0){
				fprintf(stderr,"multiProcessorCount %d\n",properties.multiProcessorCount);
				fprintf(stderr,"maxThreadsPerMultiProcessor %d\n",properties.maxThreadsPerMultiProcessor);
				
				if(properties.major == 2)
					threadsPerBlock = 192;
				else
					threadsPerBlock = 256;
			}
	}
	size_t freemem;
	size_t total;
		
	//cuda API functions always return some type of error, but if no error occured, this error is just a hipSuccess
	//errorHandler terminates program in case there was no hipSuccess reported
	errorHandler(hipMemGetInfo(&freemem, &total),__LINE__);
	
		
	fprintf(stderr,"Available memory on device: %u\n Total memory necessary on device for calculation: %u\n",freemem,totalmemsize);
		

	if(freemem < totalmemsize) {
		
				//DEBUG fprintf(stderr,"Available memory on device: %u\n Total memory necessary on device for calculation: %u\n .. splitting up calculation\n",freemem,totalmemsize);
		
		int framesRemaining = frames; //the number of frames that still have to be analysed
		int framesFinished = 0;
		int nclusters = 0; //the number of clusters already found
		int newClusters = 0;
		int *cluster = (int*)calloc(frames,sizeof(int)); //clustercenters of existing clusters
		int *devPtr_cluster; //stores the clustercenter of new found clusters
		int *devPtr_newClusters; //stores the number of new found clusters
		int nframes;	

		while(framesRemaining > 0) {
				
			size_t clust_coords_mem = nclusters * coords_size; //additional memory for the clusters' distance matrices
			errorHandler(hipMemGetInfo(&freemem, &total),__LINE__);
			
			//number of frames that fit into memory; 2MB of the total memory reported to freemem have to remain free, allocations fail otherwise (value found by trial and error)
			nframes = (freemem -2000000 - clust_dmtx_mem - sizeof(int))/(coords_size+3*sizeof(int)+sizeof(float));
			//DEBUG fprintf(stderr,"Free memory: %u, Frames remaining: %d, Frames fitting into memory: %d, Number of clusters: %d\n",freemem,framesRemaining,nframes,nclusters);
			
			//nframes is either the number of frames that fit into gpu memory, or the number of remaining frames
			if(nframes >= framesRemaining) {
				nframes = framesRemaining;
			} else {
				
				//if the number of clusters gets too high the calculation has to be stopped
				if(nclusters > nframes) {
					fprintf(stderr,"Number of clusters has exceeded number of frames that fit on GPU memory, calculation is getting too slow!\n Quitting calculation... Please choose a greater cutoff!\n");
					exit(-1);
				}
			}		
			
			//recalculate the memory sizes
			size_t memsize= nframes * coords_size;
			size_t cmemsize= nframes * sizeof(int);
			size_t dmemsize= nframes * sizeof(float);	
				
			//allocating memory on the GPU
			errorHandler(hipMalloc((void**)&devPtr_gclust_coords,memsize+clust_coords_mem),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_distance,dmemsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_frameapp1,cmemsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_frameapp2,cmemsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_cluster,cmemsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_newClusters,sizeof(int)),__LINE__);
					
			//if there were already clusters found, copy their distance matrices
			if(clust_coords_mem>0){
				//because of the overhead of a single copy instruction we prefer to copy one large data packet over lots of small ones, we use a temporary array for this
				float *clusters_coords;
				clusters_coords=(float *)malloc(clust_coords_mem);
				
				for(ii = 0; ii < nclusters; ii++)
					memcpy(clusters_coords + ii*3*nato,gclust_coords + cluster[ii]*3*nato,coords_size);
				
				errorHandler(hipMemcpy(devPtr_gclust_coords,clusters_coords,clust_coords_mem,hipMemcpyHostToDevice),__LINE__);
				free(clusters_dmtx);
			}		
			
			//copy distance matrices of the frames to gpu, they are copied right after the distance matrices of the clusters
			errorHandler(hipMemcpy(devPtr_gclust_coords + nclusters*3*nato, gclust_coords + (framesFinished + 1)*3*nato, memsize, hipMemcpyHostToDevice),__LINE__);	
				
			//Sets all indices to -1
			errorHandler(hipMemset((void*)devPtr_frameapp1,-1,cmemsize),__LINE__);
			
			//setting the distances to the cutoff for comparison on the gpu
			for(ii=framesFinished; ii<=nframes+framesFinished;ii++) {
				distance[ii] = cutoff;
			}
			errorHandler(hipMemcpy(devPtr_distance, distance+framesFinished+1, dmemsize, hipMemcpyHostToDevice),__LINE__);
	
				
			//set number of new clusters to 0
			errorHandler(hipMemset((void*)devPtr_newClusters,0,sizeof(int)),__LINE__);
					
			int blocks = nframes/threadsPerBlock +1; //in total we want 1 thread for each frame
						
			if(inp_cluster->maxspeed) {			
				//compare the frames of the chunk to the previously found clusters first
				for(ii = 0; ii < nclusters; ii++) {		
						
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					if((ii+1)%2) gRmsdClustersMax<<<blocks, threadsPerBlock>>>(nato, nframes, devPtr_gclust_coords, ii, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, nclusters, cluster[ii]);
					else gRmsdClustersMax<<<blocks, threadsPerBlock>>>(nato, nframes, devPtr_gclust_coords, ii, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, nclusters, cluster[ii]);
					
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Comparing to previous clusters %% %f\r",(double)ii/nclusters*100.0);//just a progress bar
									
				}
				
				fprintf(stderr,"\n");
				
				//then check the remaining frames against each other
				for(ii=0; ii < nframes; ii++){	
						
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					if((nclusters + ii + 1)%2) gRmsdFramesMax<<<blocks, threadsPerBlock>>>(nato, nframes, devPtr_gclust_coords,ii, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, framesFinished, nclusters, devPtr_newClusters, devPtr_cluster);
					else gRmsdFramesMax<<<blocks, threadsPerBlock>>>(nato, nframes, devPtr_gclust_coords, ii, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, framesFinished, nclusters, devPtr_newClusters, devPtr_cluster);
					
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Calculating Stage %% %f\r",(double)(framesFinished + ii)/frames*100.0);//just a progress bar
					
				}
				
			} else {	
				//compare the frames of the chunk to the previously found clusters first						
				for(ii = 0; ii < nclusters; ii++) {		
					
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					if((ii+1)%2) gRmsdClustersClosest<<<blocks, threadsPerBlock>>>(nato, nframes, devPtr_gclust_coords, ii, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, nclusters, cluster[ii]);
					else gRmsdClustersClosest<<<blocks, threadsPerBlock>>>(nato, nframes, devPtr_gclust_coords, ii, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, nclusters, cluster[ii]);
					
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Comparing to previous clusters %% %f\r",(double)ii/nclusters*100.0);//just a progress bar
					
				}
				
				
				fprintf(stderr,"\n");
				
				//then check the remaining frames against each other	
				for(ii=0; ii < nframes; ii++){
					
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					if((nclusters + ii + 1)%2) gRmsdFramesClosest<<<blocks, threadsPerBlock>>>(nato, nframes, devPtr_gclust_coords, ii, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, framesFinished, nclusters, devPtr_newClusters, devPtr_cluster);
					else gRmsdFramesClosest<<<blocks, threadsPerBlock>>>(nato, nframes, devPtr_gclust_coords, ii, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, framesFinished, nclusters, devPtr_newClusters, devPtr_cluster);
					
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Calculating Stage %% %f\r",(double)(framesFinished + ii)/frames*100.0);//just a progress bar
					
				}
				
			}
			
			printf("\n");	
			//DEBUG fprintf(stderr,"Copying to Host ..\n");
			
			//copy back to host, by adding framesFinished/nclusters to the pointers we make sure not to overwrite the results from previous runs
			if((nclusters + ii + 1)%2) errorHandler( hipMemcpy(frameapp+framesFinished + 1, devPtr_frameapp2, cmemsize, hipMemcpyDeviceToHost),__LINE__);
			else errorHandler( hipMemcpy(frameapp+framesFinished + 1, devPtr_frameapp1, cmemsize, hipMemcpyDeviceToHost),__LINE__);
			errorHandler( hipMemcpy(distance+framesFinished + 1, devPtr_distance, dmemsize, hipMemcpyDeviceToHost),__LINE__);
			errorHandler( hipMemcpy(&newClusters, devPtr_newClusters, sizeof(int), hipMemcpyDeviceToHost),__LINE__);
			errorHandler( hipMemcpy(cluster+nclusters, devPtr_cluster,newClusters*sizeof(int),hipMemcpyDeviceToHost),__LINE__);
									
			//update number of clusters and processed frames
			nclusters += newClusters;
			framesFinished += nframes;
			framesRemaining -= nframes;	
			
			//free all the GPU memory
			errorHandler(hipFree(devPtr_gclust_coords),__LINE__);
			errorHandler(hipFree(devPtr_distance),__LINE__);
			errorHandler(hipFree(devPtr_frameapp1),__LINE__);
			errorHandler(hipFree(devPtr_frameapp2),__LINE__);
			errorHandler(hipFree(devPtr_cluster),__LINE__);
			errorHandler(hipFree(devPtr_newClusters),__LINE__);			
		}	
		
			
		return 0;
	} else {
			
		//allocate gpu memory
		errorHandler(hipMalloc((void**)&devPtr_gclust_coords, memsize),__LINE__);
		errorHandler(hipMalloc((void**)&devPtr_distance, dmemsize),__LINE__);
		errorHandler(hipMalloc((void**)&devPtr_frameapp1, cmemsize),__LINE__); 
		errorHandler(hipMalloc((void**)&devPtr_frameapp2, cmemsize),__LINE__); 
						
		//copy coords to gpu
		//errorHandler(hipMemcpy(devPtr_gclust_coords, gclust_coords[0] + coords_size, frames*coords_size, hipMemcpyHostToDevice),__LINE__);
		//errorHandler(hipMemcpy(devPtr_gclust_coords+frames*nato, gclust_coords[1] + coords_size, frames*coords_size, hipMemcpyHostToDevice),__LINE__);
		//errorHandler(hipMemcpy(devPtr_gclust_coords+2*frames*nato, gclust_coords[2] + coords_size, frames*coords_size, hipMemcpyHostToDevice),__LINE__);
		errorHandler(hipMemcpy(devPtr_gclust_coords, gclust_coords + 3*nato, memsize, hipMemcpyHostToDevice),__LINE__);
		
		//setting the distances to the cutoff for comparison on the gpu
		for(ii=0; ii<=frames;ii++) {
				distance[ii] = cutoff;
			}
		errorHandler(hipMemcpy(devPtr_distance, distance+1, dmemsize, hipMemcpyHostToDevice),__LINE__);

			
		//Set all indices to -1
		errorHandler(hipMemset((void*)devPtr_frameapp1,-1,cmemsize),__LINE__);
				
	
		int blocks = frames/threadsPerBlock +1; //in total we want 1 thread for each frame
		
					
		if(inp_cluster->maxspeed) {
			for(ii=0;ii< frames;ii++){
						
				//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
				//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
				if((ii+1)%2) gRmsdMax<<<blocks, threadsPerBlock>>>(nato, frames, devPtr_gclust_coords,ii, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance);
				else gRmsdMax<<<blocks, threadsPerBlock>>>(nato, frames, devPtr_gclust_coords,ii, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance);
						
				errorHandler( hipPeekAtLastError(),__LINE__);
				fprintf(stderr,"Stage %% %f\r",(double)ii/frames*100.0);//just a progress bar
						
				}
					
			} else {
				for(ii=0;ii< frames;ii++){
						
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					if((ii+1)%2) gRmsdClosest<<<blocks, threadsPerBlock>>>(nato, frames, devPtr_gclust_coords, ii, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance);
					else gRmsdClosest<<<blocks, threadsPerBlock>>>(nato, frames, devPtr_gclust_coords, ii, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance);
						
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Stage %% %f\r",(double)ii/frames*100.0);//just a progress bar
						
				}
				
			}	
			printf("\n");
				
			//DEBUG fprintf(stderr,"Copying results to Host ..\n");
			
			if((ii+1)%2) errorHandler( hipMemcpy(frameapp+1, devPtr_frameapp2, cmemsize, hipMemcpyDeviceToHost),__LINE__);
			else errorHandler( hipMemcpy(frameapp+1, devPtr_frameapp1, cmemsize, hipMemcpyDeviceToHost),__LINE__);
				
			errorHandler( hipMemcpy(distance+1, devPtr_distance, dmemsize, hipMemcpyDeviceToHost),__LINE__);
				
			//free GPU memory
			errorHandler( hipFree(devPtr_gclust_coords),__LINE__);
			errorHandler( hipFree(devPtr_frameapp1),__LINE__);
			errorHandler( hipFree(devPtr_frameapp2),__LINE__);
			errorHandler( hipFree(devPtr_distance),__LINE__);		
			return 0;
		
	}
		
}
	

// the CUDA compiler generates C++ object files, thus the main procedure has to be an extern "C" for usage in wordom
extern "C" int gClusterDrms (struct inp_Cluster *inp_cluster,float *distance)
{
    int ii;
    float cutoff = inp_cluster->threshold;
   	int msize = inp_cluster->msize;
	int totframe = inp_cluster->totframe;
	float *gclust_dmtx = inp_cluster->gclust_dmtx;
    float nointrasegm_corr_fact = 1.0;
    int *frameapp = inp_cluster->frameapp;
    int step = inp_cluster->step;
    int frames = totframe/step+(totframe%step == 0 ? 0 : 1); //the number of frames that have to be analysed 
     
    //change correction factor if correction should be applied
	if( inp_cluster->nointrasegm != 0)
		nointrasegm_corr_fact = inp_cluster->nointrasegm_corr_fact;

	size_t dmtx_size = msize*sizeof(float); //memory size for a single distance matrix
	size_t memsize= frames * dmtx_size; //memory size for the array of distance matrices
	size_t cmemsize= frames * sizeof(int); //memory size for the frameapp array
	size_t dmemsize= frames * sizeof(float); //memory size for the distance array
	size_t totalmemsize = memsize + cmemsize + dmemsize;
	
	float *devPtr_gclust_dmtx;
	float *devPtr_distance;
	int *devPtr_frameapp1;
	int *devPtr_frameapp2;
			
	int deviceCount; // number of devices, i.e. gpus
	int device;
	int threadsPerBlock;
	struct hipDeviceProp_t properties;		
	hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
	if (cudaResultCode != hipSuccess)
		deviceCount = 0;
		
	fprintf(stderr,"Starting GPU calculation, devicecount : %d\n", deviceCount);
		
	// machines with no GPUs can still report one emulation device 	
	for (device = 0; device < deviceCount; ++device) {
		hipGetDeviceProperties(&properties, device);
		if (properties.major != 9999) // 9999 means emulation only
			if (device==0){
				fprintf(stderr,"multiProcessorCount %d\n",properties.multiProcessorCount);
				fprintf(stderr,"maxThreadsPerMultiProcessor %d\n",properties.maxThreadsPerMultiProcessor);
				
				if(properties.major == 2)
					threadsPerBlock = 192;
				else
					threadsPerBlock = 256;
			}
	}
	size_t freemem;
	size_t total;
		
	//cuda API functions always return some type of error, but if no error occured, this error is just a hipSuccess
	//errorHandler terminates program in case there was no hipSuccess reported
	errorHandler(hipMemGetInfo(&freemem, &total),__LINE__);
	
	//check if there is enough gpu memory for the job and split up the calculation if not
	if(freemem < totalmemsize) {		
		//DEBUG fprintf(stderr,"Available memory on device: %u\n Total memory necessary on device for calculation: %u\n .. splitting up calculation\n",freemem,totalmemsize);
		
		int framesRemaining = frames; //the number of frames that still have to be analysed
		int framesFinished = 0;
		int nclusters = 0; //the number of clusters already found
		int newClusters = 0;
		int *cluster = (int*)calloc(frames,sizeof(int)); //clustercenters of existing clusters
		int *devPtr_cluster; //stores the clustercenter of new found clusters
		int *devPtr_newClusters; //stores the number of new found clusters
		int nframes;	

		while(framesRemaining > 0) {
				
			size_t clust_dmtx_mem = nclusters * dmtx_size; //additional memory for the clusters' distance matrices
			errorHandler(hipMemGetInfo(&freemem, &total),__LINE__);
			
			//number of frames that fit into memory; 2MB of the total memory reported to freemem have to remain free, allocations fail otherwise (value found by trial and error)
			nframes = (freemem -2000000 - clust_dmtx_mem - sizeof(int))/(dmtx_size+3*sizeof(int)+sizeof(float));
			//DEBUG fprintf(stderr,"Free memory: %u, Frames remaining: %d, Frames fitting into memory: %d, Number of clusters: %d\n",freemem,framesRemaining,nframes,nclusters);
			
			//nframes is either the number of frames that fit into gpu memory, or the number of remaining frames
			if(nframes >= framesRemaining) {
				nframes = framesRemaining;
			} else {
				
				//if the number of clusters gets too high the calculation has to be stopped
				if(nclusters > nframes) {
					fprintf(stderr,"Number of clusters has exceeded number of frames that fit on GPU memory, calculation is getting too slow!\n Quitting calculation... Please choose a greater cutoff!\n");
					exit(-1);
				}
			}		
			
			//recalculate the memory sizes
			size_t memsize= nframes * dmtx_size;
			size_t cmemsize= nframes * sizeof(int);
			size_t dmemsize= nframes * sizeof(float);	
				
			//allocating memory on the GPU
			errorHandler(hipMalloc((void**)&devPtr_gclust_dmtx,memsize+clust_dmtx_mem),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_distance,dmemsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_frameapp1,cmemsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_frameapp2,cmemsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_cluster,cmemsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_newClusters,sizeof(int)),__LINE__);
					
			//if there were already clusters found, copy their distance matrices
			if(clust_dmtx_mem>0){
				//because of the overhead of a single copy instruction we prefer to copy one large data packet over lots of small ones, we use a temporary array for this
				float *clusters_dmtx;
				clusters_dmtx=(float *)malloc(clust_dmtx_mem);
				
				for(ii = 0; ii < nclusters; ii++)
					memcpy(clusters_dmtx + ii*msize,gclust_dmtx + cluster[ii]*msize,dmtx_size);
				
				errorHandler(hipMemcpy(devPtr_gclust_dmtx,clusters_dmtx,clust_dmtx_mem,hipMemcpyHostToDevice),__LINE__);
				free(clusters_dmtx);
			}		
			
			//copy distance matrices of the frames to gpu, they are copied right after the distance matrices of the clusters
			errorHandler(hipMemcpy(devPtr_gclust_dmtx + nclusters*msize, gclust_dmtx + (framesFinished + 1)*msize, memsize, hipMemcpyHostToDevice),__LINE__);	
				
			//Sets all indices to -1
			errorHandler(hipMemset((void*)devPtr_frameapp1,-1,cmemsize),__LINE__);
			
			//setting the distances to the cutoff for comparison on the gpu
			for(ii=framesFinished; ii<=nframes+framesFinished;ii++) {
				distance[ii] = cutoff;
			}
			errorHandler(hipMemcpy(devPtr_distance, distance+framesFinished+1, dmemsize, hipMemcpyHostToDevice),__LINE__);
				
			//set number of new clusters to 0
			errorHandler(hipMemset((void*)devPtr_newClusters,0,sizeof(int)),__LINE__);
					
			int blocks = nframes/threadsPerBlock +1; //in total we want 1 thread for each frame
						
			if(inp_cluster->maxspeed) {			
				//compare the frames of the chunk to the previously found clusters first
				for(ii = 0; ii < nclusters; ii++) {		
							
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					if((ii+1)%2) gDrmsClustersMax<<<blocks, threadsPerBlock>>>(msize, nframes, devPtr_gclust_dmtx, ii, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, nointrasegm_corr_fact, nclusters, cluster[ii]);
					else gDrmsClustersMax<<<blocks, threadsPerBlock>>>(msize, nframes, devPtr_gclust_dmtx, ii, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, nointrasegm_corr_fact, nclusters, cluster[ii]);
					
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Comparing to previous clusters %% %f\r",(double)ii/nclusters*100.0);//just a progress bar
									
				}
				
				fprintf(stderr,"\n");
				
				//then check the remaining frames against each other
				for(ii=0; ii < nframes; ii++){	
						
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					if((nclusters + ii + 1)%2) gDrmsFramesMax<<<blocks, threadsPerBlock>>>(msize, nframes, devPtr_gclust_dmtx,ii, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, nointrasegm_corr_fact, framesFinished, nclusters, devPtr_newClusters, devPtr_cluster);
					else gDrmsFramesMax<<<blocks, threadsPerBlock>>>(msize, nframes, devPtr_gclust_dmtx, ii, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, nointrasegm_corr_fact, framesFinished, nclusters, devPtr_newClusters, devPtr_cluster);
					
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Calculating Stage %% %f\r",(double)(framesFinished + ii)/frames*100.0);//just a progress bar
					
				}
				
			} else {	
				//compare the frames of the chunk to the previously found clusters first						
				for(ii = 0; ii < nclusters; ii++) {		
					
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					if((ii+1)%2) gDrmsClustersClosest<<<blocks, threadsPerBlock>>>(msize, nframes, devPtr_gclust_dmtx, ii, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, nointrasegm_corr_fact, nclusters, cluster[ii]);
					else gDrmsClustersClosest<<<blocks, threadsPerBlock>>>(msize, nframes, devPtr_gclust_dmtx, ii, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, nointrasegm_corr_fact, nclusters, cluster[ii]);
					
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Comparing to previous clusters %% %f\r",(double)ii/nclusters*100.0);//just a progress bar
					
				}
				
				fprintf(stderr,"\n");
				
				//then check the remaining frames against each other	
				for(ii=0; ii < nframes; ii++){
					
					//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
					//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
					if((nclusters + ii + 1)%2) gDrmsFramesClosest<<<blocks, threadsPerBlock>>>(msize, nframes, devPtr_gclust_dmtx, ii, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, nointrasegm_corr_fact, framesFinished, nclusters, devPtr_newClusters, devPtr_cluster);
					else gDrmsFramesClosest<<<blocks, threadsPerBlock>>>(msize, nframes, devPtr_gclust_dmtx, ii, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, nointrasegm_corr_fact, framesFinished, nclusters, devPtr_newClusters, devPtr_cluster);
					
					errorHandler( hipPeekAtLastError(),__LINE__);
					fprintf(stderr,"Calculating Stage %% %f\r",(double)(framesFinished + ii)/frames*100.0);//just a progress bar
					
				}
			}
			
			printf("\n");	
			//DEBUG fprintf(stderr,"Copying to Host ..\n");
			
			//copy back to host, by adding framesFinished/nclusters to the pointers we make sure not to overwrite the results from previous runs
			if((nclusters + ii + 1)%2) errorHandler( hipMemcpy(frameapp+framesFinished + 1, devPtr_frameapp2, cmemsize, hipMemcpyDeviceToHost),__LINE__);
			else errorHandler( hipMemcpy(frameapp+framesFinished + 1, devPtr_frameapp1, cmemsize, hipMemcpyDeviceToHost),__LINE__);
			errorHandler( hipMemcpy(distance+framesFinished + 1, devPtr_distance, dmemsize, hipMemcpyDeviceToHost),__LINE__);
			errorHandler( hipMemcpy(&newClusters, devPtr_newClusters, sizeof(int), hipMemcpyDeviceToHost),__LINE__);
			errorHandler( hipMemcpy(cluster+nclusters, devPtr_cluster,newClusters*sizeof(int),hipMemcpyDeviceToHost),__LINE__);
									
			//update number of clusters and processed frames
			nclusters += newClusters;
			framesFinished += nframes;
			framesRemaining -= nframes;	
			
			//free all the GPU memory
			errorHandler(hipFree(devPtr_gclust_dmtx),__LINE__);
			errorHandler(hipFree(devPtr_distance),__LINE__);
			errorHandler(hipFree(devPtr_frameapp1),__LINE__);
			errorHandler(hipFree(devPtr_frameapp2),__LINE__);
			errorHandler(hipFree(devPtr_cluster),__LINE__);
			errorHandler(hipFree(devPtr_newClusters),__LINE__);			
		}	
		
			
		return 0;		
		
		} else {
			
			//allocate gpu memory
			errorHandler(hipMalloc((void**)&devPtr_gclust_dmtx, memsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_distance, dmemsize),__LINE__);
			errorHandler(hipMalloc((void**)&devPtr_frameapp1, cmemsize),__LINE__); 
			errorHandler(hipMalloc((void**)&devPtr_frameapp2, cmemsize),__LINE__); 
					
			//copy distance matrices to gpu
			errorHandler(hipMemcpy(devPtr_gclust_dmtx, gclust_dmtx + msize, memsize, hipMemcpyHostToDevice),__LINE__);
		
			//Set all indices to -1
			errorHandler(hipMemset((void*)devPtr_frameapp1,-1,cmemsize),__LINE__);
			
			//setting the distances to the cutoff for comparison on the gpu
			for(ii=0; ii<=frames;ii++) {
				distance[ii] = cutoff;
			}
			errorHandler(hipMemcpy(devPtr_distance, distance+1, dmemsize, hipMemcpyHostToDevice),__LINE__);

			int blocks = frames/threadsPerBlock +1; //in total we want 1 thread for each frame
				

			for(ii=0;ii< frames;ii++){
					
				//the kernel ensures that frameapp_read has been written to frameapp_write entirely after one iteration
				//to prevent wasting time on copying frameapp_write back to frameapp_read the kernel simply gets launched with both interchanged in every second iteration
				if((ii+1)%2) gDrms<<<blocks, threadsPerBlock>>>(inp_cluster->maxspeed,msize, frames, devPtr_gclust_dmtx, ii, devPtr_frameapp1, devPtr_frameapp2, devPtr_distance, nointrasegm_corr_fact);
				else gDrms<<<blocks, threadsPerBlock>>>(inp_cluster->maxspeed,msize, frames, devPtr_gclust_dmtx, ii, devPtr_frameapp2, devPtr_frameapp1, devPtr_distance, nointrasegm_corr_fact);
					
				errorHandler( hipPeekAtLastError(),__LINE__);
				fprintf(stderr,"Stage %% %f\r",(double)ii/frames*100.0);//just a progress bar
					
			}
					
			printf("\n");
			
			//DEBUG fprintf(stderr,"Copying results to Host ..\n");
		
			if((ii+1)%2) errorHandler( hipMemcpy(frameapp+1, devPtr_frameapp2, cmemsize, hipMemcpyDeviceToHost),__LINE__);
			else errorHandler( hipMemcpy(frameapp+1, devPtr_frameapp1, cmemsize, hipMemcpyDeviceToHost),__LINE__);
			
			errorHandler( hipMemcpy(distance+1, devPtr_distance, dmemsize, hipMemcpyDeviceToHost),__LINE__);
			
			//free GPU memory
			errorHandler( hipFree(devPtr_gclust_dmtx),__LINE__);
			errorHandler( hipFree(devPtr_frameapp1),__LINE__);
			errorHandler( hipFree(devPtr_frameapp2),__LINE__);
			errorHandler( hipFree(devPtr_distance),__LINE__);		
			return 0;
	   }
	}
